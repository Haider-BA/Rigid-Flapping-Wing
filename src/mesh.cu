#include "hip/hip_runtime.h"
#include "../include/mesh.h"
#include "../include/linklist.h"
#include "../include/flow_field.h"

#include <thrust/adjacent_difference.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <iostream>
#include <fstream>


struct type_functor : public thrust::unary_function<int,int>
{
    const int IPOINT, JPOINT, KPOINT;
    
    type_functor(int _IPOINT, int _JPOINT, int _KPOINT) : IPOINT(_IPOINT), JPOINT(_JPOINT), KPOINT(_KPOINT) {}
    
    __device__
    int operator() (int INDEX) const
    {
	const int i = INDEX / (JPOINT * KPOINT);
	const int j = (INDEX % (JPOINT * KPOINT)) / KPOINT;
	const int k = (INDEX % (JPOINT * KPOINT)) % KPOINT;
	
	if( (i == 0) || (j == 0) || (k == 0) || (i == (IPOINT-1)) || (j == (JPOINT-1)) || (k == (KPOINT-1)) )
	{
	    if( ((i==0||i==IPOINT-1)&&(j==0)&&(0<=k)&&(k<=KPOINT-1))||
		((i==0||i==IPOINT-1)&&(j==JPOINT-1)&&(0<=k)&&(k<=KPOINT-1))||
		((i==0||i==IPOINT-1)&&(1<=j)&&(j<=JPOINT-2)&&(k==0))||
		((i==0||i==IPOINT-1)&&(1<=j)&&(j<=JPOINT-2)&&(k==KPOINT-1))||
		((1<=i)&&(i<=IPOINT-2)&&(j==0)&&(k==0))||
		((1<=i)&&(i<=IPOINT-2)&&(j==0)&&(k==KPOINT-1))||
		((1<=i)&&(i<=IPOINT-2)&&(j==JPOINT-1)&&(k==0))||
		((1<=i)&&(i<=IPOINT-2)&&(j==JPOINT-1)&&(k==KPOINT-1)) )
	    {
		return 6;
	    }
	    else
	    {
		return 2;
	    }
	}
	else
	{
	    return 1;
	}
    }
};

MESH_CARTESIAN :: MESH_CARTESIAN()
{
    std::cout << "Cartesian mesh is being initialized......";
	
    double Xstart, Xend, Ystart, Yend, Zstart, Zend;
    int Xsegment, Ysegment, Zsegment, XPoint = 1, YPoint = 1, ZPoint = 1;
        
    std::ifstream read;
    read.open("../flapping/cart.dat");
    if(!read.is_open()) std::cout << "Cart.dat is unable to open." << std::endl;
    read >> IPOINT >> JPOINT >> KPOINT;
    POINT_CARTESIAN = IPOINT * JPOINT * KPOINT;
    
    read >> Xstart >> Xsegment;
    double Xsegmentlength[Xsegment], Xratio[Xsegment];
    int Xsegmentpoint[Xsegment];
    Xend = Xstart;
    for(int i = 0; i < Xsegment; i++)
    {
	read >> Xsegmentlength[i] >> Xratio[i] >> Xsegmentpoint[i];
	Xend = Xend + Xsegmentlength[i];
	XPoint = XPoint + Xsegmentpoint[i];
    }
    if(XPoint != IPOINT) std::cout << "Setting of Cartesian mesh is wrong: X_direction." << std::endl;

    read >> Ystart >> Ysegment;
    double Ysegmentlength[Ysegment], Yratio[Ysegment];
    int Ysegmentpoint[Ysegment];
    Yend = Ystart;
    for(int i = 0; i < Ysegment; i++)
    {
	read >> Ysegmentlength[i] >> Yratio[i] >> Ysegmentpoint[i];
	Yend = Yend + Ysegmentlength[i];
	YPoint = YPoint + Ysegmentpoint[i];
    }
    if(YPoint != JPOINT) std::cout << "Setting of Cartesian mesh is wrong: Y_direction" << std::endl;
	
    read >> Zstart >> Zsegment;
    double Zsegmentlength[Zsegment], Zratio[Zsegment];
    int Zsegmentpoint[Zsegment];
    Zend = Zstart;
    for(int i = 0; i < Zsegment; i++)
    {
	read >> Zsegmentlength[i] >> Zratio[i] >> Zsegmentpoint[i];
	Zend = Zend + Zsegmentlength[i];
	ZPoint = ZPoint + Zsegmentpoint[i];
    }
    if(ZPoint != KPOINT) std::cout << "Setting of Cartesian mesh is wrong: Z_direction" << std::endl;
        
    read.close();
	
    XYZ[0].resize(IPOINT);
    XYZ[1].resize(JPOINT);
    XYZ[2].resize(KPOINT);
    
    DELTA[0].resize(IPOINT);
    DELTA[1].resize(JPOINT);
    DELTA[2].resize(KPOINT);
    
    VELOCITY.resize(3);
    ACCELERATION.resize(3);
    
    IINDEX.resize(IPOINT);
    JINDEX.resize(JPOINT);
    
    for(int i = 0; i < 3; i++) POINTTYPE[i].resize(POINT_CARTESIAN);
    
    COUNTTYPE.resize(POINT_CARTESIAN);  /*Need Initialization*/
    
    double tempX[IPOINT], tempY[JPOINT], tempZ[KPOINT];
    Div(Xstart, Xsegment, Xsegmentlength, Xratio, Xsegmentpoint, IPOINT, tempX);
    Div(Ystart, Ysegment, Ysegmentlength, Yratio, Ysegmentpoint, JPOINT, tempY);
    Div(Zstart, Zsegment, Zsegmentlength, Zratio, Zsegmentpoint, KPOINT, tempZ);
    thrust::copy(tempX, tempX+IPOINT, XYZ[0].begin());
    thrust::copy(tempY, tempY+JPOINT, XYZ[1].begin());
    thrust::copy(tempZ, tempZ+KPOINT, XYZ[2].begin());
    thrust::adjacent_difference(XYZ[0].begin(), XYZ[0].end(), DELTA[0].begin()); DELTA[0][0]=0;
    thrust::adjacent_difference(XYZ[1].begin(), XYZ[1].end(), DELTA[1].begin()); DELTA[1][0]=0;
    thrust::adjacent_difference(XYZ[2].begin(), XYZ[2].end(), DELTA[2].begin()); DELTA[2][0]=0;
    
    for(int i = 0; i < 3; i++)
    {
        VELOCITY[i]     = 0;
        ACCELERATION[i] = 0;
    }

    XMIN = XYZ[0][0]; XMAX = XYZ[0][IPOINT-1];
    YMIN = XYZ[1][0]; YMAX = XYZ[1][JPOINT-1];
    ZMIN = XYZ[2][0]; ZMAX = XYZ[2][KPOINT-1];
    MESHSIZE = pow( ( (XMAX-XMIN)/(IPOINT-1) * (YMAX-YMIN)/(JPOINT-1) * (ZMAX-ZMIN)/(KPOINT-1) ), 1.0/3.0 );
    SAFEDISTANCE = 0.1;

    thrust::sequence(IINDEX.begin(), IINDEX.end(), 0, KPOINT*JPOINT);
    thrust::sequence(JINDEX.begin(), JINDEX.end(), 0, KPOINT);
    
    thrust::transform(thrust::make_counting_iterator(0), 
                      thrust::make_counting_iterator(POINT_CARTESIAN),
                      POINTTYPE[0].begin(),
                      type_functor(IPOINT,JPOINT,KPOINT));
    
//    thrust::copy(POINTTYPE[0].begin(), POINTTYPE[0].end(), POINTTYPE[1].begin());
//    thrust::copy(POINTTYPE[0].begin(), POINTTYPE[0].end(), POINTTYPE[2].begin());
    POINTTYPE[1]=POINTTYPE[0]; POINTTYPE[2]=POINTTYPE[0];
    
//    cusp::print(JINDEX);
    
    
    
    std::cout << "Initialization done." << std::endl;
}

MESH_CARTESIAN :: ~MESH_CARTESIAN()
{
    std::cout << "Cartesian mesh is being deleted." << std::endl;
}

void MESH_CARTESIAN :: Div(double start, int segment, double *segmentlength, double *ratio, int *segmentpoint, int POINT, double *temp)
{
    double firstsegment;
    int offset = 1; 
    
    temp[0] = start;
    
    for(int i = 0; i < segment; i++)
    {
	if ( fabs(ratio[i] - 1 ) < 1e-4 )
	{
	    firstsegment = segmentlength[i] / segmentpoint[i];
	    
	    for (int j = 0; j < segmentpoint[i]; j++)
	    {
		start = start + firstsegment;
		
		temp[offset + j] = start;
	    }
	    
	    offset = offset + segmentpoint[i];
	}
	else
	{
	    firstsegment = segmentlength[i] * (1 - ratio[i]) / (1 - pow(ratio[i] , segmentpoint[i]));
	    
	    for (int j = 0; j < segmentpoint[i]; j++)
	    {
		start = start + firstsegment * pow(ratio[i], j);
		
		temp[offset + j] = start;
	    }
	    
	    offset = offset + segmentpoint[i];
	}
    }
}

struct is_inner
{
    __device__
    bool operator()(const int x)
    {
	return ( x == 1 );
    }
};

struct is_outer
{
    __device__
    bool operator()(const int x)
    {
	return ( x == 2 );
    }
};

MESH_LESS :: MESH_LESS( REF_FRAME& global )
{
    std::cout << "Meshless is being initialized......";

    POINT_MESSLESS = 0, INNER_POINT = 0, OUTER_POINT = 0;
    OFFSET = new int [global.obj_number];
    INNER_POINT_OFFSET = new int [global.obj_number];
    OUTER_POINT_OFFSET = new int [global.obj_number];
   
    for ( int i = 0; i < global.obj_number; i++ )
    {
	OFFSET[i] = POINT_MESSLESS;
	INNER_POINT_OFFSET[i] = INNER_POINT;
	OUTER_POINT_OFFSET[i] = OUTER_POINT;
	POINT_MESSLESS = POINT_MESSLESS + global.rigid_body[i]->POINT_NUMBER;
	INNER_POINT = INNER_POINT + global.rigid_body[i]->INNER_POINT_NUMBER;
	OUTER_POINT = OUTER_POINT + global.rigid_body[i]->OUTER_POINT_NUMBER;
	
    }
	
    for( int i = 0; i < 3; i++ )
    {
	POSITION[i].resize(POINT_MESSLESS);
	VELOCITY[i].resize(POINT_MESSLESS);
	ACCELERATION[i].resize(POINT_MESSLESS);
	POINTTYPE[i].resize(POINT_MESSLESS);
	INNER_NODE_INDEX.resize(INNER_POINT);
	OUTER_NODE_INDEX.resize(OUTER_POINT);
	SURFACE_ELE_AREA.resize(INNER_POINT);
	
	OUTER_NORMAL_VECTOR.resize(3,POINT_MESSLESS);
    }

    thrust::fill(POINTTYPE[0].begin(), POINTTYPE[0].end(), 3);
    for( int i = 0; i < global.obj_number; i++ )
    {
	for( int j = 0; j < 3; j++ )
	{
	    thrust::copy(global.rigid_body[i]->XYZ.row(j).begin(), global.rigid_body[i]->XYZ.row(j).end(), POSITION[j].begin() + OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->UVW.row(j).begin(), global.rigid_body[i]->UVW.row(j).end(), VELOCITY[j].begin() + OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->ACC.row(j).begin(), global.rigid_body[i]->ACC.row(j).end(), ACCELERATION[j].begin() + OFFSET[i]);
	    
	    thrust::copy(global.rigid_body[i]->OUTER_NORMAL_VECTOR.row(j).begin(), 
			 global.rigid_body[i]->OUTER_NORMAL_VECTOR.row(j).end(), 
			 OUTER_NORMAL_VECTOR.row(j).begin() + OFFSET[i]);
	}
	
	thrust::copy_if(thrust::make_counting_iterator(OFFSET[i]), 
			thrust::make_counting_iterator(OFFSET[i] + global.rigid_body[i]->POINT_NUMBER),
			global.rigid_body[i]->INNERMARK.begin(),
			INNER_NODE_INDEX.begin() + INNER_POINT_OFFSET[i],
			is_inner());
	
	thrust::copy_if(thrust::make_counting_iterator(OFFSET[i]),
			thrust::make_counting_iterator(OFFSET[i] + global.rigid_body[i]->POINT_NUMBER),
			global.rigid_body[i]->OUTERMARK.begin(),
			OUTER_NODE_INDEX.begin() + OUTER_POINT_OFFSET[i],
			is_outer());
	
	thrust::copy_if(global.rigid_body[i]->AREA.begin(),
			global.rigid_body[i]->AREA.end(),
			global.rigid_body[i]->INNERMARK.begin(),
			SURFACE_ELE_AREA.begin() + INNER_POINT_OFFSET[i],
			is_inner());
	
	thrust::replace_if(POINTTYPE[0].begin() + OFFSET[i],
			   POINTTYPE[0].begin() + OFFSET[i] + global.rigid_body[i]->POINT_NUMBER, 
			   global.rigid_body[i]->INNERMARK.begin(), is_inner(), 5);
    }
    POINTTYPE[1]=POINTTYPE[0]; POINTTYPE[2]=POINTTYPE[0];

    std::cout << "Initialization done." << std::endl;
}


MESH_LESS :: ~MESH_LESS()
{
    delete[] OFFSET;
    std::cout << "Meshless is being deleted." << std::endl;
}


MESH :: MESH( MESH_CARTESIAN& Cartesian, MESH_LESS& Meshless )
{
    std::cout << "Mesh is being initialized......";
    
    cartesian = &Cartesian;
    meshless  = &Meshless;

    IPOINT = cartesian->IPOINT;
    JPOINT = cartesian->JPOINT;
    KPOINT = cartesian->KPOINT;
    POINT_CARTESIAN = cartesian->POINT_CARTESIAN;
    POINT_MESSLESS = meshless->POINT_MESSLESS;
    POINT_ALL = POINT_CARTESIAN + POINT_MESSLESS;
    
    XYZ.IPOINT          = IPOINT;
    XYZ.JPOINT          = JPOINT;
    XYZ.KPOINT          = KPOINT;
    XYZ.POINT_CARTESIAN = POINT_CARTESIAN;
    XYZ.POINT_MESSLESS  = POINT_MESSLESS;
    XYZ.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
    
    UVW.IPOINT          = IPOINT;
    UVW.JPOINT          = JPOINT;
    UVW.KPOINT          = KPOINT;
    UVW.POINT_CARTESIAN = POINT_CARTESIAN;
    UVW.POINT_MESSLESS  = POINT_MESSLESS;
    UVW.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
    
    ACC.IPOINT          = IPOINT;
    ACC.JPOINT          = JPOINT;
    ACC.KPOINT          = KPOINT;
    ACC.POINT_CARTESIAN = POINT_CARTESIAN;
    ACC.POINT_MESSLESS  = POINT_MESSLESS;
    ACC.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
        
    for( int s = 0; s < 3; s++ )
    {
        XYZ.XYZ_raw[s]                   = thrust::raw_pointer_cast( cartesian->XYZ[s].data() );
        XYZ.POSITION_MESHLESS_raw[s]     = thrust::raw_pointer_cast( meshless->POSITION[s].data() );
        
        UVW.VELOCITY_MESHLESS_raw[s]     = thrust::raw_pointer_cast( meshless->VELOCITY[s].data() );
        
        ACC.ACCELERATION_MESHLESS_raw[s] = thrust::raw_pointer_cast( meshless->ACCELERATION[s].data() );
    }
    UVW.VELOCITY_CARTESIAN_raw     = thrust::raw_pointer_cast( cartesian->VELOCITY.data() );
    ACC.ACCELERATION_CARTESIAN_raw = thrust::raw_pointer_cast( cartesian->ACCELERATION.data() );
    
    std::cout << "Initialization done." << std::endl;
}

MESH :: ~MESH()
{
    std::cout << "Mesh is being deleted." << std::endl;
}

void MESH :: UPDATE_MESHLESS( REF_FRAME& global )
{
    for( int i = 0; i < global.obj_number; i++ )
    {
	for( int j = 0; j < 3; j++ )
	{
	    thrust::copy(global.rigid_body[i]->XYZ.row(j).begin(), global.rigid_body[i]->XYZ.row(j).end(), meshless->POSITION[j].begin() + meshless->OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->UVW.row(j).begin(), global.rigid_body[i]->UVW.row(j).end(), meshless->VELOCITY[j].begin() + meshless->OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->ACC.row(j).begin(), global.rigid_body[i]->ACC.row(j).end(), meshless->ACCELERATION[j].begin() + meshless->OFFSET[i]);
	    
	    thrust::copy(global.rigid_body[i]->OUTER_NORMAL_VECTOR.row(j).begin(), 
			 global.rigid_body[i]->OUTER_NORMAL_VECTOR.row(j).end(), 
			 meshless->OUTER_NORMAL_VECTOR.row(j).begin() + meshless->OFFSET[i]);
	}
    }
}

struct functor_point_reindex : public thrust::unary_function<int,int>
{
    int XLENGTH, YLENGTH, ZLENGTH, ISTART, JSTART, KSTART;
    int *IINDEX_raw, *JINDEX_raw;
    MESH_CARTESIAN   *cartesian;
    
    functor_point_reindex(int _XLENGTH, int _YLENGTH, int _ZLENGTH, int _ISTART, int _JSTART, int _KSTART, MESH_CARTESIAN *Cartesian)
    {
	XLENGTH = _XLENGTH; YLENGTH = _YLENGTH; ZLENGTH = _ZLENGTH;
	ISTART = _ISTART; JSTART = _JSTART; KSTART = _KSTART;
	
	cartesian = Cartesian;
	
	IINDEX_raw = thrust::raw_pointer_cast( cartesian->IINDEX.data() );
	JINDEX_raw = thrust::raw_pointer_cast( cartesian->JINDEX.data() );
    }
    
    __device__
    int operator()(int INDEX) const
    {
	const int i  = INDEX / (YLENGTH * ZLENGTH) + ISTART;
	const int j  = ( INDEX % (YLENGTH * ZLENGTH) ) / ZLENGTH + JSTART;
	const int k  = ( INDEX % (YLENGTH * ZLENGTH) ) % ZLENGTH + KSTART;
	
	const int ijk = IINDEX_raw[i] + JINDEX_raw[j] + k;
	
	return ijk;
    }
};

struct functor_SEARCH_TYPE0
{
    int XLENGTH, YLENGTH, ZLENGTH, ISTART, JSTART, KSTART, MLENGTH;
    MESH_CARTESIAN   *cartesian;
    MESH_LESS        *meshless;
    int              *OUTER_NODE_INDEX_raw, *IINDEX_raw, *JINDEX_raw, *POINTTYPE_raw[3];
    double           *cartesian_XYZ_raw[3], *meshless_XYZ_raw[3], *OUTER_NORMAL_VECTOR_raw;
    bool             init;
    
    functor_SEARCH_TYPE0(int _XLENGTH, int _YLENGTH, int _ZLENGTH, int _MLENGTH,
			 int _ISTART, int _JSTART, int _KSTART, 
			 MESH_CARTESIAN *Cartesian, MESH_LESS *Meshless, bool Initialization)
    {
	XLENGTH = _XLENGTH; YLENGTH = _YLENGTH; ZLENGTH = _ZLENGTH; MLENGTH = _MLENGTH;
	ISTART = _ISTART; JSTART = _JSTART; KSTART = _KSTART;
	cartesian = Cartesian; meshless = Meshless;
	init = Initialization;
	
	OUTER_NODE_INDEX_raw = thrust::raw_pointer_cast( meshless->OUTER_NODE_INDEX.data() );
	OUTER_NORMAL_VECTOR_raw = thrust::raw_pointer_cast( meshless->OUTER_NORMAL_VECTOR.values.data() );
	for(int s = 0; s < 3; s++)
	{
	    cartesian_XYZ_raw[s]       = thrust::raw_pointer_cast( cartesian->XYZ[s].data() );
	    meshless_XYZ_raw[s]        = thrust::raw_pointer_cast( meshless->POSITION[s].data() );
	    POINTTYPE_raw[s]           = thrust::raw_pointer_cast( cartesian->POINTTYPE[s].data() );
	}
	
	IINDEX_raw = thrust::raw_pointer_cast( cartesian->IINDEX.data() );
	JINDEX_raw = thrust::raw_pointer_cast( cartesian->JINDEX.data() );
    }
    
    __device__
    void operator()(int x)
    {
	const int i  = x / (YLENGTH * ZLENGTH) + ISTART;
	const int j  = ( x % (YLENGTH * ZLENGTH) ) / ZLENGTH + JSTART;
	const int k  = ( x % (YLENGTH * ZLENGTH) ) % ZLENGTH + KSTART;
	
	const int ijk = IINDEX_raw[i] + JINDEX_raw[j] + k;
	
	double sign;
	double x1, x2, y1, y2, z1, z2, d, min;
	int    im, ONI;
	
	if( POINTTYPE_raw[1][ijk] == 1 )
	{
	    
	    x1 = cartesian_XYZ_raw[0][i]; y1 = cartesian_XYZ_raw[1][j]; z1 = cartesian_XYZ_raw[2][k];
	
	    im = OUTER_NODE_INDEX_raw[0];
	    x2 = meshless_XYZ_raw[0][im]; y2 = meshless_XYZ_raw[1][im]; z2 = meshless_XYZ_raw[2][im];
	    
	    min = (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2);
	    
	    for(int s = 1; s < MLENGTH; s++)
	    {
		ONI = OUTER_NODE_INDEX_raw[s];
		x2 = meshless_XYZ_raw[0][ONI]; y2 = meshless_XYZ_raw[1][ONI]; z2 = meshless_XYZ_raw[2][ONI];
		d = (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2);
		
		if( d < min )
		{
		    min = d;
		    im = ONI;
		}
	    }
	    
	    x2 = meshless_XYZ_raw[0][im]; y2 = meshless_XYZ_raw[1][im]; z2 = meshless_XYZ_raw[2][im];
	    sign = OUTER_NORMAL_VECTOR_raw[3*im+0]*(x1-x2) + OUTER_NORMAL_VECTOR_raw[3*im+1]*(y1-y2) + OUTER_NORMAL_VECTOR_raw[3*im+2]*(z1-z2);
	    if( sign < 0 ) 
	    {
		POINTTYPE_raw[1][ijk] = 0;
		if( (POINTTYPE_raw[0][ijk] == 1) && (init == false) ) printf("Object is moving too fast. Reduce the time step size. \n");
	    }
	}
    }
};

struct functor_SEARCH_TYPE34
{
    int XLENGTH, YLENGTH, ZLENGTH, ISTART, JSTART, KSTART;
    int *IINDEX_raw, *JINDEX_raw, *POINTTYPE_raw[3];
    MESH_CARTESIAN   *cartesian;
    bool             init;
    
    functor_SEARCH_TYPE34(int _XLENGTH, int _YLENGTH, int _ZLENGTH,
			 int _ISTART, int _JSTART, int _KSTART, 
			 MESH_CARTESIAN *Cartesian, bool Initialization)
    {
	XLENGTH = _XLENGTH; YLENGTH = _YLENGTH; ZLENGTH = _ZLENGTH;
	ISTART = _ISTART; JSTART = _JSTART; KSTART = _KSTART;
	cartesian = Cartesian;
	init = Initialization;
	
	for(int s = 0; s < 3; s++)
	{
	    POINTTYPE_raw[s] = thrust::raw_pointer_cast( cartesian->POINTTYPE[s].data() );
	}
	
	IINDEX_raw = thrust::raw_pointer_cast( cartesian->IINDEX.data() );
	JINDEX_raw = thrust::raw_pointer_cast( cartesian->JINDEX.data() );
	
    }
    
    __device__
    bool operator()(int x)
    {
	const int i  = x / (YLENGTH * ZLENGTH) + ISTART;
	const int j  = ( x % (YLENGTH * ZLENGTH) ) / ZLENGTH + JSTART;
	const int k  = ( x % (YLENGTH * ZLENGTH) ) % ZLENGTH + KSTART;
	
	const int ijk = IINDEX_raw[i] + JINDEX_raw[j] + k;
	
	int ijk_w, ijk_e, ijk_s, ijk_n;
	
	bool result = false;
	
	if( POINTTYPE_raw[1][ijk] == 1 )
	{
	    ijk_w = IINDEX_raw[i-1] + JINDEX_raw[j] + k;
	    ijk_e = IINDEX_raw[i+1] + JINDEX_raw[j] + k;
	    ijk_s = IINDEX_raw[i] + JINDEX_raw[j-1] + k;
	    ijk_n = IINDEX_raw[i] + JINDEX_raw[j+1] + k;
	    
	    if( POINTTYPE_raw[1][ijk-1] == 0 || POINTTYPE_raw[1][ijk+1] == 0 || POINTTYPE_raw[1][ijk_w] == 0 ||
		POINTTYPE_raw[1][ijk_e] == 0 || POINTTYPE_raw[1][ijk_s] == 0 || POINTTYPE_raw[1][ijk_n] == 0 ) 
	    {
		if( init == true )
		{
		    POINTTYPE_raw[1][ijk] = 3;
		    result = true;
		}
		else
		{
		    if( POINTTYPE_raw[2][ijk] != 3 ) result = true;
		    if( POINTTYPE_raw[0][ijk] == 0 ) POINTTYPE_raw[1][ijk] = 4;
		    else POINTTYPE_raw[1][ijk] = 3;
		}
	    }
	}
	
	return result;
    }
};

struct functor_SEARCH_TYPE3
{
    int XLENGTH, YLENGTH, ZLENGTH, ISTART, JSTART, KSTART;
    int *IINDEX_raw, *JINDEX_raw, *POINTTYPE_raw[3];
    MESH_CARTESIAN   *cartesian;
    
    functor_SEARCH_TYPE3(int _XLENGTH, int _YLENGTH, int _ZLENGTH,
			 int _ISTART, int _JSTART, int _KSTART, 
			 MESH_CARTESIAN *Cartesian)
    {
	XLENGTH = _XLENGTH; YLENGTH = _YLENGTH; ZLENGTH = _ZLENGTH;
	ISTART = _ISTART; JSTART = _JSTART; KSTART = _KSTART;
	cartesian = Cartesian;
	
	for(int s = 0; s < 3; s++)
	{
	    POINTTYPE_raw[s] = thrust::raw_pointer_cast( cartesian->POINTTYPE[s].data() );
	}
	
	IINDEX_raw = thrust::raw_pointer_cast( cartesian->IINDEX.data() );
	JINDEX_raw = thrust::raw_pointer_cast( cartesian->JINDEX.data() );
    }
    
    __device__
    bool operator()(int x)
    {
	const int i  = x / (YLENGTH * ZLENGTH) + ISTART;
	const int j  = ( x % (YLENGTH * ZLENGTH) ) / ZLENGTH + JSTART;
	const int k  = ( x % (YLENGTH * ZLENGTH) ) % ZLENGTH + KSTART;
	
	const int ijk = IINDEX_raw[i] + JINDEX_raw[j] + k;
	
	int ijk_w, ijk_e, ijk_s, ijk_n;
	
	bool result = false;
	
	if( POINTTYPE_raw[1][ijk] == 1 )
	{
	    ijk_w = IINDEX_raw[i-1] + JINDEX_raw[j] + k;
	    ijk_e = IINDEX_raw[i+1] + JINDEX_raw[j] + k;
	    ijk_s = IINDEX_raw[i] + JINDEX_raw[j-1] + k;
	    ijk_n = IINDEX_raw[i] + JINDEX_raw[j+1] + k;
	    
	    if( POINTTYPE_raw[1][ijk-1] == 4 || POINTTYPE_raw[1][ijk+1] == 4 || POINTTYPE_raw[1][ijk_w] == 4 ||
		POINTTYPE_raw[1][ijk_e] == 4 || POINTTYPE_raw[1][ijk_s] == 4 || POINTTYPE_raw[1][ijk_n] == 4 ) 
	    {
		POINTTYPE_raw[1][ijk] = 3;
		if( POINTTYPE_raw[2][ijk] != 3 ) result = true;
	    }
	}
	
	return result;
    }
};

void MESH :: SEARCH_TYPE(bool Initialization, LINKLIST& linklist, FLOW_FIELD& flow_field)
{
    double Xmax,Ymax,Zmax,Xmin,Ymin,Zmin;
    int Istart, Iend, Jstart,Jend, Kstart, Kend;

    thrust::pair<thrust::device_vector<double>::iterator, thrust::device_vector<double>::iterator> minmax;
    
    minmax = thrust::minmax_element( meshless->POSITION[0].begin(), meshless->POSITION[0].end() );
    Xmin = *minmax.first; Xmax = *minmax.second;
    
    minmax = thrust::minmax_element( meshless->POSITION[1].begin(), meshless->POSITION[1].end() );
    Ymin = *minmax.first; Ymax = *minmax.second;
    
    minmax = thrust::minmax_element( meshless->POSITION[2].begin(), meshless->POSITION[2].end() );
    Zmin = *minmax.first; Zmax = *minmax.second;

    if( Xmin <= ( cartesian->XMIN + cartesian->SAFEDISTANCE ) || Xmax >= ( cartesian->XMAX - cartesian->SAFEDISTANCE ) ||
        Ymin <= ( cartesian->YMIN + cartesian->SAFEDISTANCE ) || Ymax >= ( cartesian->YMAX - cartesian->SAFEDISTANCE ) ||
        Zmin <= ( cartesian->ZMIN + cartesian->SAFEDISTANCE ) || Zmax >= ( cartesian->ZMAX - cartesian->SAFEDISTANCE ) )
    {
	std::cout<<"Meshless points are out of range! Need bigger box! Code terminate!"<<std::endl;
    }

    Kstart = int( (Zmin - cartesian->ZMIN)/cartesian->MESHSIZE ) - 1;
    Kend   = int( (Zmax - cartesian->ZMIN)/cartesian->MESHSIZE ) + 2;
    Jstart = int( (Ymin - cartesian->YMIN)/cartesian->MESHSIZE ) - 1;
    Jend   = int( (Ymax - cartesian->YMIN)/cartesian->MESHSIZE ) + 2;
    Istart = int( (Xmin - cartesian->XMIN)/cartesian->MESHSIZE ) - 1;
    Iend   = int( (Xmax - cartesian->XMIN)/cartesian->MESHSIZE ) + 2;

 
    thrust::for_each( thrust::make_counting_iterator(0), 
		      thrust::make_counting_iterator( (Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1) ), 
		      functor_SEARCH_TYPE0(Iend-Istart+1, Jend-Jstart+1, Kend-Kstart+1, meshless->OUTER_POINT,
					   Istart, Jstart, Kstart, 
					   cartesian, meshless, Initialization) );
  
    int Isp, Iep, Jsp, Jep, Ksp, Kep;
    Isp = Istart -1; Jsp = Jstart -1; Ksp = Kstart -1;
    Iep = Iend +1; Jep = Jend +1; Kep = Kend +1;
    
    cusp::array1d<int, cusp::device_memory> N2I((Iep-Isp+1)*(Jep-Jsp+1)*(Kep-Ksp+1));
    
    N2I.erase( thrust::copy_if( thrust::make_counting_iterator(0),
				thrust::make_counting_iterator( (Iep-Isp+1)*(Jep-Jsp+1)*(Kep-Ksp+1) ),
				N2I.begin(),
				functor_SEARCH_TYPE34(Iep-Isp+1, Jep-Jsp+1, Kep-Ksp+1, Isp, Jsp, Ksp, cartesian, Initialization) ), N2I.end() );
    
    thrust::transform( N2I.begin(), N2I.end(), N2I.begin(), functor_point_reindex(Iep-Isp+1, Jep-Jsp+1, Kep-Ksp+1, Isp, Jsp, Ksp, cartesian) );
    
    cusp::array1d<int, cusp::host_memory> N2I_host = N2I;
    
    for(int i=0; i<N2I_host.size(); i++) linklist.Insert_point(N2I_host[i]);
    
    if( Initialization == false )
    {
	N2I.resize((Iep-Isp+1)*(Jep-Jsp+1)*(Kep-Ksp+1));
    
	N2I.erase( thrust::copy_if( thrust::make_counting_iterator(0),
				    thrust::make_counting_iterator( (Iep-Isp+1)*(Jep-Jsp+1)*(Kep-Ksp+1) ),
				    N2I.begin(),
				    functor_SEARCH_TYPE3(Iep-Isp+1, Jep-Jsp+1, Kep-Ksp+1, Isp, Jsp, Ksp, cartesian) ), N2I.end() );
    
	thrust::transform( N2I.begin(), N2I.end(), N2I.begin(), functor_point_reindex(Iep-Isp+1, Jep-Jsp+1, Kep-Ksp+1, Isp, Jsp, Ksp, cartesian) );
	
	N2I_host = N2I;
	
	for(int i=0; i<N2I_host.size(); i++) linklist.Insert_point(N2I_host[i]);
    }
    
//    linklist.Delete_point(*cartesian,flow_field);
//    std::cout<<"Length of N2I is "<<linklist.LINKLISTPOINT.size()<<std::endl;
    std::cout<<Istart<<" "<<Iend<<" "<<Jstart<<" "<<Jend<<" "<<Kstart<<" "<<Kend<<std::endl;
    
    
}

struct functor_update_pointtype
{
    __device__
    int operator()(int x)
    {
	if( x == 4 ) return 3;
	else return x;
    }
};

void MESH :: UPDATE_POINTTYPE()
{
    thrust::transform( cartesian->POINTTYPE[1].begin(), cartesian->POINTTYPE[1].end(), cartesian->POINTTYPE[0].begin(), functor_update_pointtype() );
}

struct functor_update_pointtype_implicit
{
    int *POINTTYPE_raw[3];
    MESH_CARTESIAN   *cartesian;
    
    functor_update_pointtype_implicit(MESH_CARTESIAN *Cartesian)
    {
	cartesian = Cartesian;
	
	for(int s = 0; s < 3; s++)
	{
	    POINTTYPE_raw[s] = thrust::raw_pointer_cast( cartesian->POINTTYPE[s].data() );
	}
    }
    __device__
    void operator()(int x)
    {
	int ijk = x;
	
	if( POINTTYPE_raw[1][ijk] == 4 )
	{
	    POINTTYPE_raw[2][ijk] = 3;
	}
	else
	{
	    POINTTYPE_raw[2][ijk] = POINTTYPE_raw[1][ijk];
	}
	if ( (POINTTYPE_raw[1][ijk] != 2) && (POINTTYPE_raw[1][ijk] != 6) ) POINTTYPE_raw[1][ijk] = 1;
	
//	if ( POINTTYPE_raw[2][ijk] == 3 && (POINTTYPE_raw[1][ijk]==0||POINTTYPE_raw[1][ijk]==1) ) printf("Point %d is needed to delete.\n",ijk);
    }
};

void MESH :: UPDATE_POINTTYPE_IMPLICIT()
{
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(cartesian->POINT_CARTESIAN), functor_update_pointtype_implicit(cartesian) );
}

void MESH :: TEST_FUNCTION()
{
    for(int i=0; i<cartesian->POINT_CARTESIAN; i++)
    {
	if( cartesian->POINTTYPE[2][i]==3 && (cartesian->POINTTYPE[1][i]==0||cartesian->POINTTYPE[1][i]==1) ) std::cout<<"Point "<<i<<" is needed to delete. IN TEST"<<std::endl;
    }
}

