
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <omp.h>

bool GPU_detection()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
	return false;
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }
    
    int dev;
    for (dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("Device %d: \"%s\"  |  ", dev, deviceProp.name);
    }
    std::cout << std::endl << std::endl;
    
    //manually specfiy the GPU device.
    dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    std::cout << "Set CUDA device(s) to " << deviceProp.name << "." << std::endl << std::endl;

    return true;
}

void CPU_detection()
{
    int No_CPU = omp_get_num_procs();
    
    omp_set_num_threads(No_CPU);
    
    std::cout << "Set OpenMP thread(s) to " << No_CPU << "." << std::endl << std::endl;
}