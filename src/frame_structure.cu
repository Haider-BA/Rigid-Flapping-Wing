#include "hip/hip_runtime.h"
#include "../include/frame_structure.h"

#include <iostream>
#include <fstream>
#include <math.h>

#include <cusp/blas/blas.h>


double REF_FRAME :: compute_angular_velocity1(double Time)
{
    return -compute_theta(Time,1)*sin( compute_phi(Time,0) ) + compute_psi(Time,1)*cos( compute_theta(Time,0) )*cos( compute_phi(Time,0) ) ;
}

double REF_FRAME :: compute_angular_velocity2(double Time)
{
    return compute_theta(Time,1)*cos( compute_phi(Time,0) ) + compute_psi(Time,1)*cos( compute_theta(Time,0) )*sin( compute_phi(Time,0) ) ;
}

double REF_FRAME :: compute_angular_velocity3(double Time)
{
    return compute_phi(Time,1) - compute_psi(Time,1)*sin( compute_theta(Time,0) ) ;
}

double REF_FRAME :: compute_angular_acceleration1(double Time)
{
    return (  - compute_theta(Time,2) * sin( compute_phi(Time,0) )
    		  - compute_theta(Time,1) * compute_phi(Time,1) * cos( compute_phi(Time,0) )
              + compute_psi(Time,2) * cos( compute_theta(Time,0) ) * cos( compute_phi(Time,0) )
              - compute_psi(Time,1) * compute_theta(Time,1) * sin( compute_theta(Time,0) )*cos( compute_phi(Time,0) )
              - compute_psi(Time,1) * compute_phi(Time,1) * cos( compute_theta(Time,0) )*sin( compute_phi(Time,0))  );
}

double REF_FRAME :: compute_angular_acceleration2(double Time)
{
    return (  compute_theta(Time,2) * cos( compute_phi(Time,0) )
              - compute_theta(Time,1) * compute_phi(Time,1) * sin( compute_phi(Time,0) )
              + compute_psi(Time,2) * cos(compute_theta(Time,0)) * sin( compute_phi(Time,0) )
              - compute_psi(Time,1) * compute_theta(Time,1) * sin( compute_theta(Time,0) )*sin( compute_phi(Time,0) )
              + compute_psi(Time,1) * compute_phi(Time,1) * cos( compute_theta(Time,0) )*cos( compute_phi(Time,0) )  );
}

double REF_FRAME :: compute_angular_acceleration3(double Time)
{
    return (  compute_phi(Time,2)
              - compute_psi(Time,1) * compute_theta(Time,1) * cos( compute_theta(Time,0) )
              - compute_psi(Time,2) * sin( compute_theta(Time,0) )  );
}

void REF_FRAME :: update_all()
{
    position[0]=compute_x(Time,0); //std::cout<<"x "<<position[0]<<" ";
    position[1]=compute_y(Time,0); //std::cout<<"y "<<position[1]<<" ";
    position[2]=compute_z(Time,0); //std::cout<<"z "<<position[2]<<std::endl;
    velocity[0]=compute_x(Time,1);
    velocity[1]=compute_y(Time,1);
    velocity[2]=compute_z(Time,1);
    acceleration[0]=compute_x(Time,2);
    acceleration[1]=compute_y(Time,2);
    acceleration[2]=compute_z(Time,2);
    
    angle[0]=compute_phi(Time,0); //std::cout<<"phi = "<<angle[0]<<" ";
    angle[1]=compute_theta(Time,0); //std::cout<<"theta = "<<angle[1]<<" ";
    angle[2]=compute_psi(Time,0); //std::cout<<"psi = "<<angle[2]<<std::endl;
    angular_velocity[0]=compute_angular_velocity1(Time);
    angular_velocity[1]=compute_angular_velocity2(Time);
    angular_velocity[2]=compute_angular_velocity3(Time);
    compute_orientation();
    angular_acceleration[0]=compute_angular_acceleration1(Time);
    angular_acceleration[1]=compute_angular_acceleration2(Time);
    angular_acceleration[2]=compute_angular_acceleration3(Time);
}


void REF_FRAME :: copy_new2old()
{
    for(int i=0;i<3;i++)
    {
	position_old[i]             =    position[i];
	velocity_old[i]             =    velocity[i];
	acceleration_old[i]         =    acceleration[i];
	angle_old[i]                =    angle[i];
	angular_velocity_old[i]     =    angular_velocity[i];
	angular_acceleration_old[i] =    angular_acceleration[i];
	force_old[i]                =    force[i];
	torque_old[i]               =    torque[i];
	momentum_old[i]             =    momentum[i];
	angular_momentum_old[i]     =    angular_momentum[i];
	for(int j=0;j<3;j++)
	{
	    orientation_old[i][j]   =    orientation[i][j];
	    inertia_old[i][j]       =    inertia[i][j];
	}
    }
}

void REF_FRAME :: compute_orientation_tmp()
{
	double Phi,Theta,Psi;
	Phi=angle_tmp[0];Theta=angle_tmp[1];Psi=angle_tmp[2];
	orientation_tmp[0][0] = cos(Phi)*cos(Theta);
	orientation_tmp[0][1] = cos(Phi)*sin(Theta)*sin(Psi) - sin(Phi)*cos(Psi);
	orientation_tmp[0][2] = cos(Phi)*sin(Theta)*cos(Psi) + sin(Phi)*sin(Psi);
	orientation_tmp[1][0] = sin(Phi)*cos(Theta);
	orientation_tmp[1][1] = sin(Phi)*sin(Theta)*sin(Psi) + cos(Phi)*cos(Psi);
	orientation_tmp[1][2] = sin(Phi)*sin(Theta)*cos(Psi) - cos(Phi)*sin(Psi);
	orientation_tmp[2][0] = -sin(Theta);
	orientation_tmp[2][1] = cos(Theta)*sin(Psi);
	orientation_tmp[2][2] = cos(Theta)*cos(Psi);
}

void REF_FRAME :: compute_orientation()
{
	double Phi,Theta,Psi;
	Phi=angle[0];Theta=angle[1];Psi=angle[2];
	orientation[0][0] = cos(Phi)*cos(Theta);
	orientation[0][1] = cos(Phi)*sin(Theta)*sin(Psi) - sin(Phi)*cos(Psi);
	orientation[0][2] = cos(Phi)*sin(Theta)*cos(Psi) + sin(Phi)*sin(Psi);
	orientation[1][0] = sin(Phi)*cos(Theta);
	orientation[1][1] = sin(Phi)*sin(Theta)*sin(Psi) + cos(Phi)*cos(Psi);
	orientation[1][2] = sin(Phi)*sin(Theta)*cos(Psi) - cos(Phi)*sin(Psi);
	orientation[2][0] = -sin(Theta);
	orientation[2][1] = cos(Theta)*sin(Psi);
	orientation[2][2] = cos(Theta)*cos(Psi);
}

void REF_FRAME :: memory_allocate_for_rigid_body(RIGID_OBJ *temp)
{    
    temp->XYZ.resize(3,temp->POINT_NUMBER);
    temp->UVW.resize(3,temp->POINT_NUMBER);
    temp->ACC.resize(3,temp->POINT_NUMBER);
    temp->OUTER_NORMAL_VECTOR.resize(3,temp->POINT_NUMBER);
    
    temp->TRIANGLE.resize(3,temp->TRIANGLE_NUMBER);
    temp->TETRAHEDRON.resize(4,temp->TETRAHEDRON_NUMBER);

    temp->AREA.resize(temp->POINT_NUMBER);
    temp->INNERMARK.resize(temp->POINT_NUMBER);
    temp->OUTERMARK.resize(temp->POINT_NUMBER);
}

RIGID_OBJ* REF_FRAME :: get_rigid_body_from_file(const char *filename)
{
    RIGID_OBJ *temp = new RIGID_OBJ;
    std::ifstream in;
    in.open(filename);
    in>>temp->IDENTITY;
    in>>temp->LEVEL;
    in>>temp->POINTMODE;
    in>>temp->POINT_NUMBER>>temp->TETRAHEDRON_NUMBER>>temp->TRIANGLE_NUMBER;
    in>>temp->INNER_POINT_NUMBER>>temp->OUTER_POINT_NUMBER;

    memory_allocate_for_rigid_body(temp);

    double *readXYZ[3], *readONV[3], *readAREA;
    int    *readINNERMARK, *readOUTERMARK, *readTETRAHEDRON[4], *readTRIANGLE[3];
    
    for(int i = 0; i < 3; i++) readXYZ[i] = new double [temp->POINT_NUMBER];
    for(int i = 0; i < 3; i++) readONV[i] = new double [temp->POINT_NUMBER];
    readAREA = new double [temp->POINT_NUMBER];
    readINNERMARK = new int [temp->POINT_NUMBER];
    readOUTERMARK = new int [temp->POINT_NUMBER];
    for(int i = 0; i < 4; i++) readTETRAHEDRON[i] = new int [temp->TETRAHEDRON_NUMBER];
    for(int i = 0; i < 3; i++) readTRIANGLE[i] = new int [temp->TRIANGLE_NUMBER];
    
    for (int i = 0; i < temp->POINT_NUMBER; i++)
    {
	in >> readXYZ[0][i] >> readXYZ[1][i] >> readXYZ[2][i];
	in >> readONV[0][i] >> readONV[1][i] >> readONV[2][i];
	in >> readINNERMARK[i] >> readOUTERMARK[i];
	in >> readAREA[i];
	
//	std::cout<<"Read file, onv is "<<readONV[0][i]<<" "<<readONV[1][i]<<" "<<readONV[2][i]<<" "<<sqrt(readONV[0][i]*readONV[0][i]+readONV[1][i]*readONV[1][i]+readONV[2][i]*readONV[2][i])<<std::endl;
    }
    for (int j = 0; j < temp->TETRAHEDRON_NUMBER; j++)
    {
	in >> readTETRAHEDRON[0][j] >> readTETRAHEDRON[1][j] >> readTETRAHEDRON[2][j] >> readTETRAHEDRON[3][j];
    }
    for (int j = 0; j < temp->TRIANGLE_NUMBER; j++)
    {
	in >> readTRIANGLE[0][j] >> readTRIANGLE[1][j] >> readTRIANGLE[2][j];
    }
    in.close();
    
    for(int i = 0; i < 3; i++) thrust::copy(readXYZ[i], readXYZ[i] + temp->POINT_NUMBER, temp->XYZ.row(i).begin());
    for(int i = 0; i < 3; i++) thrust::fill(temp->UVW.row(i).begin(), temp->UVW.row(i).end(), 0.0);
    for(int i = 0; i < 3; i++) thrust::fill(temp->ACC.row(i).begin(), temp->ACC.row(i).end(), 0.0);
    for(int i = 0; i < 3; i++) thrust::copy(readONV[i], readONV[i] + temp->POINT_NUMBER, temp->OUTER_NORMAL_VECTOR.row(i).begin());
    thrust::copy(readINNERMARK, readINNERMARK + temp->POINT_NUMBER, temp->INNERMARK.begin());
    thrust::copy(readOUTERMARK, readOUTERMARK + temp->POINT_NUMBER, temp->OUTERMARK.begin());
    thrust::copy(readAREA, readAREA + temp->POINT_NUMBER, temp->AREA.begin());
    for(int i = 0; i < 4; i++) thrust::copy(readTETRAHEDRON[i], readTETRAHEDRON[i] + temp->TETRAHEDRON_NUMBER, temp->TETRAHEDRON.row(i).begin());
    for(int i = 0; i < 3; i++) thrust::copy(readTRIANGLE[i], readTRIANGLE[i] + temp->TRIANGLE_NUMBER, temp->TRIANGLE.row(i).begin());
    
    for(int i = 0; i < 3; i++) delete[] readXYZ[i];
    for(int i = 0; i < 3; i++) delete[] readONV[i];
    delete[] readAREA;
    delete[] readINNERMARK;
    delete[] readOUTERMARK;
    for(int i = 0; i < 4; i++) delete[] readTETRAHEDRON[i];
    for(int i = 0; i < 3; i++) delete[] readTRIANGLE[i];
    
    std::cout << "Point is: " << temp->POINT_NUMBER << std::endl;

    return temp;
}


RIGID_OBJ* REF_FRAME :: get_rigid_body_from_subframe(int no_fra,int no_obj)
{
    if(subframe_number==0){std::cout<<"There is no subframe!"<<std::endl;}
    if(no_fra<0||no_fra>=subframe_number){std::cout<<"The range of no_fra is wrong!"<<std::endl;}
    if(no_obj<0||no_obj>=sub[no_fra]->obj_number){std::cout<<"The range of no_obj is wrong!"<<std::endl;}
    RIGID_OBJ *temp=new RIGID_OBJ;
    temp->IDENTITY=sub[no_fra]->rigid_body[no_obj]->IDENTITY;
    temp->LEVEL=sub[no_fra]->rigid_body[no_obj]->LEVEL-1;
    temp->POINTMODE=sub[no_fra]->rigid_body[no_obj]->POINTMODE;
    temp->POINT_NUMBER=sub[no_fra]->rigid_body[no_obj]->POINT_NUMBER;
    temp->TETRAHEDRON_NUMBER=sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON_NUMBER;
    temp->TRIANGLE_NUMBER=sub[no_fra]->rigid_body[no_obj]->TRIANGLE_NUMBER;
    temp->INNER_POINT_NUMBER=sub[no_fra]->rigid_body[no_obj]->INNER_POINT_NUMBER;
    temp->OUTER_POINT_NUMBER=sub[no_fra]->rigid_body[no_obj]->OUTER_POINT_NUMBER;

    memory_allocate_for_rigid_body(temp);

    cusp::array2d<double,cusp::device_memory,cusp::column_major> T_t(3,3), Omega(3,3), Omega_T(3,3), Omega_Omega_T(3,3), dOmega(3,3), dOmega_T(3,3), Omega_Omega_T_plus_dOmega_T(3,3);
//    double T_t[3][3],Omega[3][3],Omega_T[3][3],Omega_Omega_T[3][3],dOmega[3][3],dOmega_T[3][3];
/*
    T_t[0][0]=sub[no_fra]->orientation[0][0];
    T_t[0][1]=sub[no_fra]->orientation[0][1];
    T_t[0][2]=sub[no_fra]->orientation[0][2];
    T_t[1][0]=sub[no_fra]->orientation[1][0];
    T_t[1][1]=sub[no_fra]->orientation[1][1];
    T_t[1][2]=sub[no_fra]->orientation[1][2];
    T_t[2][0]=sub[no_fra]->orientation[2][0];
    T_t[2][1]=sub[no_fra]->orientation[2][1];
    T_t[2][2]=sub[no_fra]->orientation[2][2];

    Omega[0][0]=0;
    Omega[0][1]=-sub[no_fra]->angular_velocity[2];
    Omega[0][2]=sub[no_fra]->angular_velocity[1];
    Omega[1][0]=sub[no_fra]->angular_velocity[2];
    Omega[1][1]=0;
    Omega[1][2]=-sub[no_fra]->angular_velocity[0];
    Omega[2][0]=-sub[no_fra]->angular_velocity[1]; 
    Omega[2][1]=sub[no_fra]->angular_velocity[0];   
    Omega[2][2]=0;

    Multiply(Omega,T_t,Omega_T);
    Multiply(Omega,Omega_T,Omega_Omega_T);

    dOmega[0][0]=0;
    dOmega[0][1]=-sub[no_fra]->angular_acceleration[2];  
    dOmega[0][2]=sub[no_fra]->angular_acceleration[1];
    dOmega[1][0]=sub[no_fra]->angular_acceleration[2];  
    dOmega[1][1]=0;                                      
    dOmega[1][2]=-sub[no_fra]->angular_acceleration[0];
    dOmega[2][0]=-sub[no_fra]->angular_acceleration[1]; 
    dOmega[2][1]=sub[no_fra]->angular_acceleration[0];
    dOmega[2][2]=0;

    Multiply(dOmega,T_t,dOmega_T);
*/
    T_t(0,0)=sub[no_fra]->orientation[0][0];
    T_t(0,1)=sub[no_fra]->orientation[0][1];
    T_t(0,2)=sub[no_fra]->orientation[0][2];
    T_t(1,0)=sub[no_fra]->orientation[1][0];
    T_t(1,1)=sub[no_fra]->orientation[1][1];
    T_t(1,2)=sub[no_fra]->orientation[1][2];
    T_t(2,0)=sub[no_fra]->orientation[2][0];
    T_t(2,1)=sub[no_fra]->orientation[2][1];
    T_t(2,2)=sub[no_fra]->orientation[2][2];

    Omega(0,0)=0;
    Omega(0,1)=-sub[no_fra]->angular_velocity[2];
    Omega(0,2)=sub[no_fra]->angular_velocity[1];
    Omega(1,0)=sub[no_fra]->angular_velocity[2];
    Omega(1,1)=0;
    Omega(1,2)=-sub[no_fra]->angular_velocity[0];
    Omega(2,0)=-sub[no_fra]->angular_velocity[1]; 
    Omega(2,1)=sub[no_fra]->angular_velocity[0];   
    Omega(2,2)=0;

    cusp::blas::gemm(Omega,T_t,Omega_T);
    cusp::blas::gemm(Omega,Omega_T,Omega_Omega_T);
    
    dOmega(0,0)=0;
    dOmega(0,1)=-sub[no_fra]->angular_acceleration[2];  
    dOmega(0,2)=sub[no_fra]->angular_acceleration[1];
    dOmega(1,0)=sub[no_fra]->angular_acceleration[2];  
    dOmega(1,1)=0;                                      
    dOmega(1,2)=-sub[no_fra]->angular_acceleration[0];
    dOmega(2,0)=-sub[no_fra]->angular_acceleration[1]; 
    dOmega(2,1)=sub[no_fra]->angular_acceleration[0];
    dOmega(2,2)=0;

    cusp::blas::gemm(dOmega,T_t,dOmega_T);
    
    
    cusp::array2d<double, cusp::device_memory, cusp::column_major> sub_position(3,temp->POINT_NUMBER), sub_velocity(3,temp->POINT_NUMBER), sub_acceleration(3,temp->POINT_NUMBER), TEMP(3,temp->POINT_NUMBER);
    
    for(int i = 0; i < 3; i++){
	thrust::fill(sub_position.row(i).begin(), sub_position.row(i).end(), sub[no_fra]->position[i]);
	thrust::fill(sub_velocity.row(i).begin(), sub_velocity.row(i).end(), sub[no_fra]->velocity[i]);
	thrust::fill(sub_acceleration.row(i).begin(), sub_acceleration.row(i).end(), sub[no_fra]->acceleration[i]);
    }

    
    
    cusp::blas::gemm(T_t,sub[no_fra]->rigid_body[no_obj]->XYZ,temp->XYZ);
    cusp::blas::geam(temp->XYZ,sub_position,temp->XYZ,1.0,1.0);
    
    cusp::blas::gemm(T_t,sub[no_fra]->rigid_body[no_obj]->UVW,temp->UVW);
    cusp::blas::gemm(Omega_T,sub[no_fra]->rigid_body[no_obj]->XYZ,TEMP);
    cusp::blas::geam(temp->UVW,TEMP,temp->UVW,1.0,1.0);
    cusp::blas::geam(temp->UVW,sub_velocity,temp->UVW,1.0,1.0);
    
    cusp::blas::geam(Omega_Omega_T,dOmega_T,Omega_Omega_T_plus_dOmega_T,1.0,1.0);
    cusp::blas::gemm(T_t,sub[no_fra]->rigid_body[no_obj]->ACC,temp->ACC);
    cusp::blas::gemm(Omega_T,sub[no_fra]->rigid_body[no_obj]->UVW,TEMP);
    cusp::blas::geam(temp->ACC,TEMP,temp->ACC,1.0,2.0);
    cusp::blas::gemm(Omega_Omega_T_plus_dOmega_T,sub[no_fra]->rigid_body[no_obj]->XYZ,TEMP);
    cusp::blas::geam(temp->ACC,TEMP,temp->ACC,1.0,1.0);
    cusp::blas::geam(temp->ACC,sub_acceleration,temp->ACC,1.0,1.0);
    
    cusp::blas::gemm(T_t,sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR,temp->OUTER_NORMAL_VECTOR);
    
    
    temp->AREA = sub[no_fra]->rigid_body[no_obj]->AREA;
    temp->INNERMARK = sub[no_fra]->rigid_body[no_obj]->INNERMARK;
    temp->OUTERMARK = sub[no_fra]->rigid_body[no_obj]->OUTERMARK;
    
    temp->TETRAHEDRON = sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON;
    temp->TRIANGLE = sub[no_fra]->rigid_body[no_obj]->TRIANGLE;
    
/*    
    for (int s=0; s<temp->POINT_NUMBER; s++)
	{
		temp->XYZ[0][s] = sub[no_fra]->position[0]
                        + T_t[0][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + T_t[0][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + T_t[0][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		temp->XYZ[1][s] = sub[no_fra]->position[1]
                        + T_t[1][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + T_t[1][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + T_t[1][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		temp->XYZ[2][s] = sub[no_fra]->position[2]
                        + T_t[2][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + T_t[2][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + T_t[2][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		
		temp->UVW[0][s] = sub[no_fra]->velocity[0]
                        + T_t[0][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + T_t[0][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + T_t[0][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + Omega_T[0][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + Omega_T[0][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + Omega_T[0][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		temp->UVW[1][s] = sub[no_fra]->velocity[1]
                        + T_t[1][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + T_t[1][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + T_t[1][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + Omega_T[1][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + Omega_T[1][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + Omega_T[1][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		temp->UVW[2][s] = sub[no_fra]->velocity[2]
                        + T_t[2][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + T_t[2][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + T_t[2][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + Omega_T[2][0]*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + Omega_T[2][1]*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + Omega_T[2][2]*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		
		temp->ACC[0][s] = sub[no_fra]->acceleration[0]
                        + T_t[0][0]*sub[no_fra]->rigid_body[no_obj]->ACC[0][s]
                        + T_t[0][1]*sub[no_fra]->rigid_body[no_obj]->ACC[1][s]
                        + T_t[0][2]*sub[no_fra]->rigid_body[no_obj]->ACC[2][s]
                        + 2*Omega_T[0][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + 2*Omega_T[0][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + 2*Omega_T[0][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + (Omega_Omega_T[0][0]+dOmega_T[0][0])*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + (Omega_Omega_T[0][1]+dOmega_T[0][1])*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + (Omega_Omega_T[0][2]+dOmega_T[0][2])*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];	    
		temp->ACC[1][s] = sub[no_fra]->acceleration[1]
                        + T_t[1][0]*sub[no_fra]->rigid_body[no_obj]->ACC[0][s]
                        + T_t[1][1]*sub[no_fra]->rigid_body[no_obj]->ACC[1][s]
                        + T_t[1][2]*sub[no_fra]->rigid_body[no_obj]->ACC[2][s]
                        + 2*Omega_T[1][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + 2*Omega_T[1][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + 2*Omega_T[1][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + (Omega_Omega_T[1][0]+dOmega_T[1][0])*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + (Omega_Omega_T[1][1]+dOmega_T[1][1])*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + (Omega_Omega_T[1][2]+dOmega_T[1][2])*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		temp->ACC[2][s] = sub[no_fra]->acceleration[2]
                        + T_t[2][0]*sub[no_fra]->rigid_body[no_obj]->ACC[0][s]
                        + T_t[2][1]*sub[no_fra]->rigid_body[no_obj]->ACC[1][s]
                        + T_t[2][2]*sub[no_fra]->rigid_body[no_obj]->ACC[2][s]
                        + 2*Omega_T[2][0]*sub[no_fra]->rigid_body[no_obj]->UVW[0][s]
                        + 2*Omega_T[2][1]*sub[no_fra]->rigid_body[no_obj]->UVW[1][s]
                        + 2*Omega_T[2][2]*sub[no_fra]->rigid_body[no_obj]->UVW[2][s]
                        + (Omega_Omega_T[2][0]+dOmega_T[2][0])*sub[no_fra]->rigid_body[no_obj]->XYZ[0][s]
                        + (Omega_Omega_T[2][1]+dOmega_T[2][1])*sub[no_fra]->rigid_body[no_obj]->XYZ[1][s]
                        + (Omega_Omega_T[2][2]+dOmega_T[2][2])*sub[no_fra]->rigid_body[no_obj]->XYZ[2][s];
		
		temp->OUTER_NORMAL_VECTOR[0][s] = 
                          T_t[0][0]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[0][s]
                        + T_t[0][1]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[1][s]
                        + T_t[0][2]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[2][s];
		temp->OUTER_NORMAL_VECTOR[1][s] = 
                          T_t[1][0]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[0][s]
                        + T_t[1][1]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[1][s]
                        + T_t[1][2]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[2][s];
		temp->OUTER_NORMAL_VECTOR[2][s] = 
                          T_t[2][0]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[0][s]
                        + T_t[2][1]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[1][s]
                        + T_t[2][2]*sub[no_fra]->rigid_body[no_obj]->OUTER_NORMAL_VECTOR[2][s];
		
		temp->AREA[s]=sub[no_fra]->rigid_body[no_obj]->AREA[s];
		
		temp->INNERMARK[s]=sub[no_fra]->rigid_body[no_obj]->INNERMARK[s];
		temp->OUTERMARK[s]=sub[no_fra]->rigid_body[no_obj]->OUTERMARK[s];
	}
	
	
	for (int j=0; j<temp->TETRAHEDRON_NUMBER; j++)
	{
		temp->TETRAHEDRON[0][j]=sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON[0][j];
		temp->TETRAHEDRON[1][j]=sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON[1][j];
		temp->TETRAHEDRON[2][j]=sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON[2][j];
		temp->TETRAHEDRON[3][j]=sub[no_fra]->rigid_body[no_obj]->TETRAHEDRON[3][j];
	}
	
	for (int j=0; j<temp->TRIANGLE_NUMBER; j++)
	{
		temp->TRIANGLE[0][j]=sub[no_fra]->rigid_body[no_obj]->TRIANGLE[0][j];
		temp->TRIANGLE[1][j]=sub[no_fra]->rigid_body[no_obj]->TRIANGLE[1][j];
		temp->TRIANGLE[2][j]=sub[no_fra]->rigid_body[no_obj]->TRIANGLE[2][j];
	}
*/	
	
	return temp;
}
/*
void Ref_Frame :: change_ref_frame()
{
    if(subframe_number==0){cout<<"No sub frame exist!"<<endl;}
    if(obj_number==0){cout<<"No object exist in current frame!"<<endl;}
    double T_t[3][3],Omega[3][3],Omega_T[3][3],Omega_Omega_T[3][3],dOmega[3][3],dOmega_T[3][3];
	
    for(int k=0;k<obj_number;k++)
    {
	for(int i=0;i<subframe_number;i++)
	{
	    if(sub[i]->obj_number==0){cout<<"No object exist in sub frame!"<<endl;continue;}
	
	    T_t[0][0]=sub[i]->orientation[0][0];
	    T_t[0][1]=sub[i]->orientation[0][1];
	    T_t[0][2]=sub[i]->orientation[0][2];
	    T_t[1][0]=sub[i]->orientation[1][0];
	    T_t[1][1]=sub[i]->orientation[1][1];
	    T_t[1][2]=sub[i]->orientation[1][2];
	    T_t[2][0]=sub[i]->orientation[2][0];
	    T_t[2][1]=sub[i]->orientation[2][1];
	    T_t[2][2]=sub[i]->orientation[2][2];
			
	    Omega[0][0]=0;
	    Omega[0][1]=-sub[i]->angular_velocity[2];  
	    Omega[0][2]=sub[i]->angular_velocity[1];
	    Omega[1][0]=sub[i]->angular_velocity[2];  
	    Omega[1][1]=0;
	    Omega[1][2]=-sub[i]->angular_velocity[0];
	    Omega[2][0]=-sub[i]->angular_velocity[1]; 
	    Omega[2][1]=sub[i]->angular_velocity[0];   
	    Omega[2][2]=0;
			
	    Multiply(Omega,T_t,Omega_T);
			
	    Multiply(Omega,Omega_T,Omega_Omega_T);
			
	    dOmega[0][0]=0;
	    dOmega[0][1]=-sub[i]->angular_acceleration[2];  
	    dOmega[0][2]=sub[i]->angular_acceleration[1];
	    dOmega[1][0]=sub[i]->angular_acceleration[2];  
	    dOmega[1][1]=0;
	    dOmega[1][2]=-sub[i]->angular_acceleration[0];
	    dOmega[2][0]=-sub[i]->angular_acceleration[1]; 
	    dOmega[2][1]=sub[i]->angular_acceleration[0];
	    dOmega[2][2]=0;
			
	    Multiply(dOmega,T_t,dOmega_T);
			
	    for(int j=0;j<sub[i]->obj_number;j++)
	    {
		if((rigid_body[k]->IDENTITY==sub[i]->rigid_body[j]->IDENTITY)&&(rigid_body[k]->LEVEL==sub[i]->rigid_body[j]->LEVEL-1))
		{
		    for(int s=0;s<rigid_body[k]->POINT_NUMBER;s++)
		    {
			rigid_body[k]->XYZ[0][s] = sub[i]->position[0]
                        + T_t[0][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + T_t[0][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + T_t[0][2]*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->XYZ[1][s] = sub[i]->position[1]
                        + T_t[1][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + T_t[1][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + T_t[1][2]*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->XYZ[2][s] = sub[i]->position[2]
                        + T_t[2][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + T_t[2][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + T_t[2][2]*sub[i]->rigid_body[j]->XYZ[2][s];
						
			rigid_body[k]->UVW[0][s] = sub[i]->velocity[0]
                        + T_t[0][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        + T_t[0][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        + T_t[0][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        + Omega_T[0][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + Omega_T[0][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + Omega_T[0][2]*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->UVW[1][s] = sub[i]->velocity[1]
                        + T_t[1][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        + T_t[1][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        + T_t[1][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        + Omega_T[1][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + Omega_T[1][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + Omega_T[1][2]*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->UVW[2][s] = sub[i]->velocity[2]
                        + T_t[2][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        + T_t[2][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        + T_t[2][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        + Omega_T[2][0]*sub[i]->rigid_body[j]->XYZ[0][s]
                        + Omega_T[2][1]*sub[i]->rigid_body[j]->XYZ[1][s]
                        + Omega_T[2][2]*sub[i]->rigid_body[j]->XYZ[2][s];
						
			rigid_body[k]->ACC[0][s] = sub[i]->acceleration[0]
                        + T_t[0][0]*sub[i]->rigid_body[j]->ACC[0][s]
                        + T_t[0][1]*sub[i]->rigid_body[j]->ACC[1][s]
                        + T_t[0][2]*sub[i]->rigid_body[j]->ACC[2][s]
                        + 2*Omega_T[0][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        + 2*Omega_T[0][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        + 2*Omega_T[0][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        + (Omega_Omega_T[0][0]+dOmega_T[0][0])*sub[i]->rigid_body[j]->XYZ[0][s]
                        + (Omega_Omega_T[0][1]+dOmega_T[0][1])*sub[i]->rigid_body[j]->XYZ[1][s]
                        + (Omega_Omega_T[0][2]+dOmega_T[0][2])*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->ACC[1][s] = sub[i]->acceleration[1]
                        +T_t[1][0]*sub[i]->rigid_body[j]->ACC[0][s]
                        +T_t[1][1]*sub[i]->rigid_body[j]->ACC[1][s]
                        +T_t[1][2]*sub[i]->rigid_body[j]->ACC[2][s]
                        +2*Omega_T[1][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        +2*Omega_T[1][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        +2*Omega_T[1][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        +(Omega_Omega_T[1][0]+dOmega_T[1][0])*sub[i]->rigid_body[j]->XYZ[0][s]
                        +(Omega_Omega_T[1][1]+dOmega_T[1][1])*sub[i]->rigid_body[j]->XYZ[1][s]
                        +(Omega_Omega_T[1][2]+dOmega_T[1][2])*sub[i]->rigid_body[j]->XYZ[2][s];
			rigid_body[k]->ACC[2][s]=sub[i]->acceleration[2]
                        +T_t[2][0]*sub[i]->rigid_body[j]->ACC[0][s]
                        +T_t[2][1]*sub[i]->rigid_body[j]->ACC[1][s]
                        +T_t[2][2]*sub[i]->rigid_body[j]->ACC[2][s]
                        +2*Omega_T[2][0]*sub[i]->rigid_body[j]->UVW[0][s]
                        +2*Omega_T[2][1]*sub[i]->rigid_body[j]->UVW[1][s]
                        +2*Omega_T[2][2]*sub[i]->rigid_body[j]->UVW[2][s]
                        +(Omega_Omega_T[2][0]+dOmega_T[2][0])*sub[i]->rigid_body[j]->XYZ[0][s]
                        +(Omega_Omega_T[2][1]+dOmega_T[2][1])*sub[i]->rigid_body[j]->XYZ[1][s]
                        +(Omega_Omega_T[2][2]+dOmega_T[2][2])*sub[i]->rigid_body[j]->XYZ[2][s];
													  
			rigid_body[k]->OUTER_NORMAL_VECTOR[0][s] = 
                          T_t[0][0]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[0][s]
                        + T_t[0][1]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[1][s]
                        + T_t[0][2]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[2][s];
			rigid_body[k]->OUTER_NORMAL_VECTOR[1][s] = 
                          T_t[1][0]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[0][s]
                        + T_t[1][1]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[1][s]
                        + T_t[1][2]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[2][s];
			rigid_body[k]->OUTER_NORMAL_VECTOR[2][s] = 
			              T_t[2][0]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[0][s]
			            + T_t[2][1]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[1][s]
			            + T_t[2][2]*sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR[2][s];

		    }
		}
	    }
	}
    }
}
*/

void REF_FRAME :: change_ref_frame()
{
    if(subframe_number==0){std::cout<<"No sub frame exist!"<<std::endl;}
    if(obj_number==0){std::cout<<"No object exist in current frame!"<<std::endl;}

    cusp::array2d<double,cusp::device_memory,cusp::column_major> T_t(3,3), Omega(3,3), Omega_T(3,3), Omega_Omega_T(3,3), dOmega(3,3), dOmega_T(3,3), Omega_Omega_T_plus_dOmega_T(3,3);
    
    for(int k=0;k<obj_number;k++)
    {
	for(int i=0;i<subframe_number;i++)
	{
	    if(sub[i]->obj_number==0){std::cout<<"No object exist in sub frame!"<<std::endl;continue;}

	    T_t(0,0)=sub[i]->orientation[0][0];
	    T_t(0,1)=sub[i]->orientation[0][1];
	    T_t(0,2)=sub[i]->orientation[0][2];
	    T_t(1,0)=sub[i]->orientation[1][0];
	    T_t(1,1)=sub[i]->orientation[1][1];
	    T_t(1,2)=sub[i]->orientation[1][2];
	    T_t(2,0)=sub[i]->orientation[2][0];
	    T_t(2,1)=sub[i]->orientation[2][1];
	    T_t(2,2)=sub[i]->orientation[2][2];

	    Omega(0,0)=0;
	    Omega(0,1)=-sub[i]->angular_velocity[2];
	    Omega(0,2)=sub[i]->angular_velocity[1];
	    Omega(1,0)=sub[i]->angular_velocity[2];
	    Omega(1,1)=0;
	    Omega(1,2)=-sub[i]->angular_velocity[0];
	    Omega(2,0)=-sub[i]->angular_velocity[1]; 
	    Omega(2,1)=sub[i]->angular_velocity[0];   
	    Omega(2,2)=0;

	    cusp::blas::gemm(Omega,T_t,Omega_T);
	    cusp::blas::gemm(Omega,Omega_T,Omega_Omega_T);

	    dOmega(0,0)=0;
	    dOmega(0,1)=-sub[i]->angular_acceleration[2];  
	    dOmega(0,2)=sub[i]->angular_acceleration[1];
	    dOmega(1,0)=sub[i]->angular_acceleration[2];  
	    dOmega(1,1)=0;                                      
	    dOmega(1,2)=-sub[i]->angular_acceleration[0];
	    dOmega(2,0)=-sub[i]->angular_acceleration[1]; 
	    dOmega(2,1)=sub[i]->angular_acceleration[0];
	    dOmega(2,2)=0;
	    
	    cusp::blas::gemm(dOmega,T_t,dOmega_T);
	    cusp::array2d<double, cusp::device_memory, cusp::column_major> sub_position(3,rigid_body[k]->POINT_NUMBER), sub_velocity(3,rigid_body[k]->POINT_NUMBER), sub_acceleration(3,rigid_body[k]->POINT_NUMBER), TEMP(3,rigid_body[k]->POINT_NUMBER);

	    for(int j=0;j<sub[i]->obj_number;j++)
	    {
		if((rigid_body[k]->IDENTITY==sub[i]->rigid_body[j]->IDENTITY)&&(rigid_body[k]->LEVEL==sub[i]->rigid_body[j]->LEVEL-1))
		{
		    for(int p = 0; p < 3; p++){
			thrust::fill(sub_position.row(p).begin(), sub_position.row(p).end(), sub[i]->position[p]);
			thrust::fill(sub_velocity.row(p).begin(), sub_velocity.row(p).end(), sub[i]->velocity[p]);
			thrust::fill(sub_acceleration.row(p).begin(), sub_acceleration.row(p).end(), sub[i]->acceleration[p]);
		    }
		    
		    cusp::blas::gemm(T_t,sub[i]->rigid_body[j]->XYZ,rigid_body[k]->XYZ);
		    cusp::blas::geam(rigid_body[k]->XYZ,sub_position,rigid_body[k]->XYZ,1.0,1.0);
		    
		    cusp::blas::gemm(T_t,sub[i]->rigid_body[j]->UVW,rigid_body[k]->UVW);
		    cusp::blas::gemm(Omega_T,sub[i]->rigid_body[j]->XYZ,TEMP);
		    cusp::blas::geam(rigid_body[k]->UVW,TEMP,rigid_body[k]->UVW,1.0,1.0);
		    cusp::blas::geam(rigid_body[k]->UVW,sub_velocity,rigid_body[k]->UVW,1.0,1.0);
		    
		    cusp::blas::geam(Omega_Omega_T,dOmega_T,Omega_Omega_T_plus_dOmega_T,1.0,1.0);
		    cusp::blas::gemm(T_t,sub[i]->rigid_body[j]->ACC,rigid_body[k]->ACC);
		    cusp::blas::gemm(Omega_T,sub[i]->rigid_body[j]->UVW,TEMP);
		    cusp::blas::geam(rigid_body[k]->ACC,TEMP,rigid_body[k]->ACC,1.0,2.0);
		    cusp::blas::gemm(Omega_Omega_T_plus_dOmega_T,sub[i]->rigid_body[j]->XYZ,TEMP);
		    cusp::blas::geam(rigid_body[k]->ACC,TEMP,rigid_body[k]->ACC,1.0,1.0);
		    cusp::blas::geam(rigid_body[k]->ACC,sub_acceleration,rigid_body[k]->ACC,1.0,1.0);
		    
		    cusp::blas::gemm(T_t,sub[i]->rigid_body[j]->OUTER_NORMAL_VECTOR,rigid_body[k]->OUTER_NORMAL_VECTOR);
		}
	    }
	}
    }
}

