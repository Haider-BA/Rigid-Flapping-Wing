#include "hip/hip_runtime.h"
#include "../include/linklist.h"
#include <thrust/remove.h>
#include <thrust/for_each.h>
#include <iostream>
#include <stdio.h>

LINKLIST :: LINKLIST()
{
    std::cout << "Linklist is being initialized......";
    
    std::cout << "Initialization done." << std::endl;
}

LINKLIST :: ~LINKLIST()
{
    std::cout << "Linklist is being deleted." << std::endl;
}

void LINKLIST :: Insert_point(int i)
{
    /*maybe add filter later*/
    LINKLIST_MEMBER temp;
    temp.Meshless_Ind = i;
    LINKLISTPOINT.push_back(temp);
}

struct is_del
{
    int *TYPE1_raw, *TYPE2_raw, *TYPET_raw;
    double *U_raw, *V_raw, *W_raw, *P_raw;
    
    is_del(MESH_CARTESIAN *cartesian, FLOW_FIELD& flow_field)
    {
        TYPE1_raw = thrust::raw_pointer_cast( cartesian->POINTTYPE[0].data() );
        TYPE2_raw = thrust::raw_pointer_cast( cartesian->POINTTYPE[1].data() );
        TYPET_raw = thrust::raw_pointer_cast( cartesian->POINTTYPE[2].data() );
	
        U_raw = thrust::raw_pointer_cast( &flow_field.U.row(0)[0] );
        V_raw = thrust::raw_pointer_cast( &flow_field.V.row(0)[0] );
        W_raw = thrust::raw_pointer_cast( &flow_field.W.row(0)[0] );
        P_raw = thrust::raw_pointer_cast( &flow_field.P.row(0)[0] );
    }
    
    __device__
    bool operator()(const LINKLIST_MEMBER& x)
    {
        int i = x.Meshless_Ind;
	
	bool s = ( TYPE2_raw[i] == 0 || TYPE2_raw[i] == 1 ) && (TYPET_raw[i] == 3);
	if (s)
	{
	    printf("Delete point %d activated.\n", i);
	    if (TYPE2_raw[i] == 0)
	    {
		U_raw[i] = 0.0; V_raw[i] = 0.0; W_raw[i] = 0.0; P_raw[i] = 0.0;
	    }
	}
	
        return s;
    }
};
/*
void LINKLIST :: Delete_point(MESH_CARTESIAN& cartesian, FLOW_FIELD& flow_field)
{
    int i;
    for(thrust::device_vector<LINKLIST_MEMBER>::iterator iter = LINKLISTPOINT.begin(); iter != LINKLISTPOINT.end(); iter++)  
    {
	i = (static_cast<LINKLIST_MEMBER>(*iter)).Meshless_Ind;
	
	if( cartesian.POINTTYPE[2][i]==3 && (cartesian.POINTTYPE[1][i]==0||cartesian.POINTTYPE[1][i]==1) ) std::cout<<"Point "<<i<<" is needed to delete. IN LINKLIST"<<std::endl;
    }
  
  
    std::cout<<"Before Deleting point, the length is "<<LINKLISTPOINT.size()<<" ";
    LINKLISTPOINT.erase( thrust::remove_if( LINKLISTPOINT.begin(),
                                            LINKLISTPOINT.end(),
                                            is_del( cartesian, flow_field ) ), LINKLISTPOINT.end() );
    std::cout<<"After deleting point, the length is "<<LINKLISTPOINT.size()<<std::endl;
}
*/


void LINKLIST :: Delete_point(MESH& mesh, FLOW_FIELD& flow_field)
{
    std::cout<<"Before Deleting point, the length is "<<LINKLISTPOINT.size()<<" ";
    LINKLISTPOINT.erase( thrust::remove_if( LINKLISTPOINT.begin(),
                                            LINKLISTPOINT.end(),
                                            is_del( mesh.cartesian, flow_field ) ), LINKLISTPOINT.end() );
    std::cout<<"After deleting point, the length is "<<LINKLISTPOINT.size()<<std::endl;
}

struct is_fresh
{
    double *U_raw, *V_raw, *W_raw, *P_raw, *U_old_raw, *V_old_raw, *W_old_raw, *P_old_raw;
    int    *TYPE2_raw;
    MESH::UNIFIED_POSITION XYZ_raw;
    
    is_fresh(MESH& mesh, MESH_CARTESIAN& cartesian, FLOW_FIELD& flow_field)
    {        
        XYZ_raw = mesh.XYZ;
        
        U_raw = thrust::raw_pointer_cast( &flow_field.U.row(0)[0] );
        V_raw = thrust::raw_pointer_cast( &flow_field.V.row(0)[0] );
        W_raw = thrust::raw_pointer_cast( &flow_field.W.row(0)[0] );
        P_raw = thrust::raw_pointer_cast( &flow_field.P.row(0)[0] );
        
        U_old_raw = thrust::raw_pointer_cast( &flow_field.U_old.row(0)[0] );
        V_old_raw = thrust::raw_pointer_cast( &flow_field.V_old.row(0)[0] );
        W_old_raw = thrust::raw_pointer_cast( &flow_field.W_old.row(0)[0] );
        P_old_raw = thrust::raw_pointer_cast( &flow_field.P_old.row(0)[0] );
	
        TYPE2_raw = thrust::raw_pointer_cast( cartesian.POINTTYPE[1].data() );
    }
    
    __device__
    void operator()( LINKLIST_MEMBER& s )
    {
        int ijk = s.Meshless_Ind, ijk_p, i, io, j;
        
        if( TYPE2_raw[ijk] == 4 )
        {
	    double tmp1, tmp2, tmp3, tmp4, tmp5, tmp6, tmp7, tmp8, tmp9;
	    double tmp1_u, tmp2_u, tmp3_u, tmp4_u, tmp5_u, tmp6_u, tmp7_u, tmp8_u, tmp9_u;
	    double tmp1_v, tmp2_v, tmp3_v, tmp4_v, tmp5_v, tmp6_v, tmp7_v, tmp8_v, tmp9_v;
	    double tmp1_w, tmp2_w, tmp3_w, tmp4_w, tmp5_w, tmp6_w, tmp7_w, tmp8_w, tmp9_w;
	    double tmp1_p, tmp2_p, tmp3_p, tmp4_p, tmp5_p, tmp6_p, tmp7_p, tmp8_p, tmp9_p;
	    double drp[2], dx[2], dy[2], dz[2], tmpu[2], tmpv[2], tmpw[2], tmpp[2]; 
	    double dr;
	  
	    for(i = 0; i < 2; i++)
	    {
		ijk_p = s.Nb_Points[i];

		dx[i] = XYZ_raw(0, ijk_p) - XYZ_raw(0, ijk);
		dy[i] = XYZ_raw(1, ijk_p) - XYZ_raw(1, ijk);
		dz[i] = XYZ_raw(2, ijk_p) - XYZ_raw(2, ijk);
		drp[i] = sqrt(dx[i]*dx[i]+dy[i]*dy[i]+dz[i]*dz[i]);

		tmp1_u=0;tmp2_u=0;tmp3_u=0;tmp4_u=0;tmp5_u=0;tmp6_u=0;tmp7_u=0;tmp8_u=0;tmp9_u=0;
		tmp1_v=0;tmp2_v=0;tmp3_v=0;tmp4_v=0;tmp5_v=0;tmp6_v=0;tmp7_v=0;tmp8_v=0;tmp9_v=0;
		tmp1_w=0;tmp2_w=0;tmp3_w=0;tmp4_w=0;tmp5_w=0;tmp6_w=0;tmp7_w=0;tmp8_w=0;tmp9_w=0;
		tmp1_p=0;tmp2_p=0;tmp3_p=0;tmp4_p=0;tmp5_p=0;tmp6_p=0;tmp7_p=0;tmp8_p=0;tmp9_p=0;
		tmp1=0;tmp2=0;tmp3=0;tmp4=0;tmp5=0;tmp6=0;tmp7=0;tmp8=0;tmp9=0;
		
		for(io = 0; io < NB; io++)
		{
		    j = s.Nb_Points[io];
		    
		    tmp1_u = tmp1_u + s.Csvd[0][io]*U_raw[j];
		    tmp2_u = tmp2_u + s.Csvd[1][io]*U_raw[j];
		    tmp3_u = tmp3_u + s.Csvd[2][io]*U_raw[j];
		    tmp4_u = tmp4_u + s.Csvd[3][io]*U_raw[j];
		    tmp5_u = tmp5_u + s.Csvd[4][io]*U_raw[j];
		    tmp6_u = tmp6_u + s.Csvd[5][io]*U_raw[j];
		    tmp7_u = tmp7_u + s.Csvd[6][io]*U_raw[j];
		    tmp8_u = tmp8_u + s.Csvd[7][io]*U_raw[j];
		    tmp9_u = tmp9_u + s.Csvd[8][io]*U_raw[j];
		    
		    tmp1_v = tmp1_v + s.Csvd[0][io]*V_raw[j];
		    tmp2_v = tmp2_v + s.Csvd[1][io]*V_raw[j];
		    tmp3_v = tmp3_v + s.Csvd[2][io]*V_raw[j];
		    tmp4_v = tmp4_v + s.Csvd[3][io]*V_raw[j];
		    tmp5_v = tmp5_v + s.Csvd[4][io]*V_raw[j];
		    tmp6_v = tmp6_v + s.Csvd[5][io]*V_raw[j];
		    tmp7_v = tmp7_v + s.Csvd[6][io]*V_raw[j];
		    tmp8_v = tmp8_v + s.Csvd[7][io]*V_raw[j];
		    tmp9_v = tmp9_v + s.Csvd[8][io]*V_raw[j];
		
		    tmp1_w = tmp1_w + s.Csvd[0][io]*W_raw[j];
		    tmp2_w = tmp2_w + s.Csvd[1][io]*W_raw[j];
		    tmp3_w = tmp3_w + s.Csvd[2][io]*W_raw[j];
		    tmp4_w = tmp4_w + s.Csvd[3][io]*W_raw[j];
		    tmp5_w = tmp5_w + s.Csvd[4][io]*W_raw[j];
		    tmp6_w = tmp6_w + s.Csvd[5][io]*W_raw[j];
		    tmp7_w = tmp7_w + s.Csvd[6][io]*W_raw[j];
		    tmp8_w = tmp8_w + s.Csvd[7][io]*W_raw[j];
		    tmp9_w = tmp9_w + s.Csvd[8][io]*W_raw[j];
		
		    tmp1_p = tmp1_p + s.Csvd[0][io]*P_raw[j];
		    tmp2_p = tmp2_p + s.Csvd[1][io]*P_raw[j];
		    tmp3_p = tmp3_p + s.Csvd[2][io]*P_raw[j];
		    tmp4_p = tmp4_p + s.Csvd[3][io]*P_raw[j];
		    tmp5_p = tmp5_p + s.Csvd[4][io]*P_raw[j];
		    tmp6_p = tmp6_p + s.Csvd[5][io]*P_raw[j];
		    tmp7_p = tmp7_p + s.Csvd[6][io]*P_raw[j];
		    tmp8_p = tmp8_p + s.Csvd[7][io]*P_raw[j];
		    tmp9_p = tmp9_p + s.Csvd[8][io]*P_raw[j];
					
		    tmp1+=s.Csvd[0][io];
		    tmp2+=s.Csvd[1][io];
		    tmp3+=s.Csvd[2][io];
		    tmp4+=s.Csvd[3][io];
		    tmp5+=s.Csvd[4][io];
		    tmp6+=s.Csvd[5][io];
		    tmp7+=s.Csvd[6][io];
		    tmp8+=s.Csvd[7][io];
		    tmp9+=s.Csvd[8][io];
		}
		
		
		dr=(1-dx[i]*tmp1-dy[i]*tmp2-dz[i]*tmp3-0.5*dx[i]*dx[i]*tmp4-0.5*dy[i]*dy[i]*tmp5-0.5*dz[i]*dz[i]*tmp6-dx[i]*dy[i]*tmp7-dx[i]*dz[i]*tmp8-dy[i]*dz[i]*tmp9); 
		tmpu[i]=(U_raw[ijk_p]-dx[i]*tmp1_u-dy[i]*tmp2_u-dz[i]*tmp3_u-0.5*dx[i]*dx[i]*tmp4_u
			-0.5*dy[i]*dy[i]*tmp5_u-0.5*dz[i]*dz[i]*tmp6_u-dx[i]*dy[i]*tmp7_u
			-dx[i]*dz[i]*tmp8_u-dy[i]*dz[i]*tmp9_u)/dr;
		tmpv[i]=(V_raw[ijk_p]-dx[i]*tmp1_v-dy[i]*tmp2_v-dz[i]*tmp3_v-0.5*dx[i]*dx[i]*tmp4_v
			-0.5*dy[i]*dy[i]*tmp5_v-0.5*dz[i]*dz[i]*tmp6_v-dx[i]*dy[i]*tmp7_v
			-dx[i]*dz[i]*tmp8_v-dy[i]*dz[i]*tmp9_v)/dr;
		tmpw[i]=(W_raw[ijk_p]-dx[i]*tmp1_w-dy[i]*tmp2_w-dz[i]*tmp3_w-0.5*dx[i]*dx[i]*tmp4_w
			-0.5*dy[i]*dy[i]*tmp5_w-0.5*dz[i]*dz[i]*tmp6_w-dx[i]*dy[i]*tmp7_w
			-dx[i]*dz[i]*tmp8_w-dy[i]*dz[i]*tmp9_w)/dr;
		tmpp[i]=(P_raw[ijk_p]-dx[i]*tmp1_p-dy[i]*tmp2_p-dz[i]*tmp3_p-0.5*dx[i]*dx[i]*tmp4_p
			-0.5*dy[i]*dy[i]*tmp5_p-0.5*dz[i]*dz[i]*tmp6_p-dx[i]*dy[i]*tmp7_p
			-dx[i]*dz[i]*tmp8_p-dy[i]*dz[i]*tmp9_p)/dr;
			

	    }
			
	    U_raw[ijk] = (drp[1]*tmpu[0]+drp[0]*tmpu[1])/(drp[0]+drp[1]);
	    V_raw[ijk] = (drp[1]*tmpv[0]+drp[0]*tmpv[1])/(drp[0]+drp[1]);
	    W_raw[ijk] = (drp[1]*tmpw[0]+drp[0]*tmpw[1])/(drp[0]+drp[1]);
	    P_raw[ijk] = (drp[1]*tmpp[0]+drp[0]*tmpp[1])/(drp[0]+drp[1]);
	    U_old_raw[ijk] = U_raw[ijk];
	    V_old_raw[ijk] = V_raw[ijk];
	    W_old_raw[ijk] = W_raw[ijk];
	    P_old_raw[ijk] = P_raw[ijk];
        }
    }
};

void LINKLIST :: Fresh_point(MESH& mesh, MESH_CARTESIAN& cartesian, FLOW_FIELD& flow_field)
{
    thrust::for_each( LINKLISTPOINT.begin(), LINKLISTPOINT.end(), is_fresh( mesh, cartesian, flow_field ) );
}

struct is_update
{
    double *U_raw, *V_raw, *W_raw, *P_raw, *U_old_raw, *V_old_raw, *W_old_raw, *P_old_raw;
    int    *TYPE2_raw, *TYPET_raw;
    MESH::UNIFIED_POSITION XYZ_raw;
    
    is_update(MESH& mesh, MESH_CARTESIAN& cartesian, FLOW_FIELD& flow_field)
    {
        XYZ_raw = mesh.XYZ;
        
        U_raw = thrust::raw_pointer_cast( &flow_field.U.row(0)[0] );
        V_raw = thrust::raw_pointer_cast( &flow_field.V.row(0)[0] );
        W_raw = thrust::raw_pointer_cast( &flow_field.W.row(0)[0] );
        P_raw = thrust::raw_pointer_cast( &flow_field.P.row(0)[0] );
        
        U_old_raw = thrust::raw_pointer_cast( &flow_field.U_old.row(0)[0] );
        V_old_raw = thrust::raw_pointer_cast( &flow_field.V_old.row(0)[0] );
        W_old_raw = thrust::raw_pointer_cast( &flow_field.W_old.row(0)[0] );
        P_old_raw = thrust::raw_pointer_cast( &flow_field.P_old.row(0)[0] );
        
        TYPE2_raw = thrust::raw_pointer_cast( cartesian.POINTTYPE[1].data() );
        TYPET_raw = thrust::raw_pointer_cast( cartesian.POINTTYPE[2].data() );
    }
    
    __device__
    void operator()( LINKLIST_MEMBER& s )
    {
        int ijk = s.Meshless_Ind, ijk_p, i, io, j;
        
        if( (TYPE2_raw[ijk] == 1 && TYPET_raw[ijk] == 0) ||
            (TYPE2_raw[ijk] == 3 && TYPET_raw[ijk] == 0) ||
            (TYPE2_raw[ijk] == 4 && TYPET_raw[ijk] == 3) )
        {
            double tmp1, tmp2, tmp3, tmp4, tmp5, tmp6, tmp7, tmp8, tmp9;
            double tmp1_u, tmp2_u, tmp3_u, tmp4_u, tmp5_u, tmp6_u, tmp7_u, tmp8_u, tmp9_u;
            double tmp1_v, tmp2_v, tmp3_v, tmp4_v, tmp5_v, tmp6_v, tmp7_v, tmp8_v, tmp9_v;
            double tmp1_w, tmp2_w, tmp3_w, tmp4_w, tmp5_w, tmp6_w, tmp7_w, tmp8_w, tmp9_w;
            double tmp1_p, tmp2_p, tmp3_p, tmp4_p, tmp5_p, tmp6_p, tmp7_p, tmp8_p, tmp9_p;
            double drp[2], dx[2], dy[2], dz[2], tmpu[2], tmpv[2], tmpw[2], tmpp[2];
            double dr;
            
            for(i = 0; i < 2; i++)
            {
                ijk_p = s.Nb_Points[i];
                
                dx[i] = XYZ_raw(0, ijk_p) - XYZ_raw(0, ijk);
                dy[i] = XYZ_raw(1, ijk_p) - XYZ_raw(1, ijk);
                dz[i] = XYZ_raw(2, ijk_p) - XYZ_raw(2, ijk);
                drp[i] = sqrt(dx[i]*dx[i]+dy[i]*dy[i]+dz[i]*dz[i]);

                tmp1_u=0;tmp2_u=0;tmp3_u=0;tmp4_u=0;tmp5_u=0;tmp6_u=0;tmp7_u=0;tmp8_u=0;tmp9_u=0;
                tmp1_v=0;tmp2_v=0;tmp3_v=0;tmp4_v=0;tmp5_v=0;tmp6_v=0;tmp7_v=0;tmp8_v=0;tmp9_v=0;
                tmp1_w=0;tmp2_w=0;tmp3_w=0;tmp4_w=0;tmp5_w=0;tmp6_w=0;tmp7_w=0;tmp8_w=0;tmp9_w=0;
                tmp1_p=0;tmp2_p=0;tmp3_p=0;tmp4_p=0;tmp5_p=0;tmp6_p=0;tmp7_p=0;tmp8_p=0;tmp9_p=0;
                tmp1=0;tmp2=0;tmp3=0;tmp4=0;tmp5=0;tmp6=0;tmp7=0;tmp8=0;tmp9=0;
                
                for(io = 0; io < NB; io++)
                {
                    j = s.Nb_Points[io];
                    
                    tmp1_u = tmp1_u + s.Csvd[0][io]*U_raw[j];
                    tmp2_u = tmp2_u + s.Csvd[1][io]*U_raw[j];
                    tmp3_u = tmp3_u + s.Csvd[2][io]*U_raw[j];
                    tmp4_u = tmp4_u + s.Csvd[3][io]*U_raw[j];
                    tmp5_u = tmp5_u + s.Csvd[4][io]*U_raw[j];
                    tmp6_u = tmp6_u + s.Csvd[5][io]*U_raw[j];
                    tmp7_u = tmp7_u + s.Csvd[6][io]*U_raw[j];
                    tmp8_u = tmp8_u + s.Csvd[7][io]*U_raw[j];
                    tmp9_u = tmp9_u + s.Csvd[8][io]*U_raw[j];
                    
                    tmp1_v = tmp1_v + s.Csvd[0][io]*V_raw[j];
                    tmp2_v = tmp2_v + s.Csvd[1][io]*V_raw[j];
                    tmp3_v = tmp3_v + s.Csvd[2][io]*V_raw[j];
                    tmp4_v = tmp4_v + s.Csvd[3][io]*V_raw[j];
                    tmp5_v = tmp5_v + s.Csvd[4][io]*V_raw[j];
                    tmp6_v = tmp6_v + s.Csvd[5][io]*V_raw[j];
                    tmp7_v = tmp7_v + s.Csvd[6][io]*V_raw[j];
                    tmp8_v = tmp8_v + s.Csvd[7][io]*V_raw[j];
                    tmp9_v = tmp9_v + s.Csvd[8][io]*V_raw[j];
                    
                    tmp1_w = tmp1_w + s.Csvd[0][io]*W_raw[j];
                    tmp2_w = tmp2_w + s.Csvd[1][io]*W_raw[j];
                    tmp3_w = tmp3_w + s.Csvd[2][io]*W_raw[j];
                    tmp4_w = tmp4_w + s.Csvd[3][io]*W_raw[j];
                    tmp5_w = tmp5_w + s.Csvd[4][io]*W_raw[j];
                    tmp6_w = tmp6_w + s.Csvd[5][io]*W_raw[j];
                    tmp7_w = tmp7_w + s.Csvd[6][io]*W_raw[j];
                    tmp8_w = tmp8_w + s.Csvd[7][io]*W_raw[j];
                    tmp9_w = tmp9_w + s.Csvd[8][io]*W_raw[j];
                    
                    tmp1_p = tmp1_p + s.Csvd[0][io]*P_raw[j];
                    tmp2_p = tmp2_p + s.Csvd[1][io]*P_raw[j];
                    tmp3_p = tmp3_p + s.Csvd[2][io]*P_raw[j];
                    tmp4_p = tmp4_p + s.Csvd[3][io]*P_raw[j];
                    tmp5_p = tmp5_p + s.Csvd[4][io]*P_raw[j];
                    tmp6_p = tmp6_p + s.Csvd[5][io]*P_raw[j];
                    tmp7_p = tmp7_p + s.Csvd[6][io]*P_raw[j];
                    tmp8_p = tmp8_p + s.Csvd[7][io]*P_raw[j];
                    tmp9_p = tmp9_p + s.Csvd[8][io]*P_raw[j];
                    
                    tmp1+=s.Csvd[0][io];
                    tmp2+=s.Csvd[1][io];
                    tmp3+=s.Csvd[2][io];
                    tmp4+=s.Csvd[3][io];
                    tmp5+=s.Csvd[4][io];
                    tmp6+=s.Csvd[5][io];
                    tmp7+=s.Csvd[6][io];
                    tmp8+=s.Csvd[7][io];
                    tmp9+=s.Csvd[8][io];
                }
                
                
                dr=(1-dx[i]*tmp1-dy[i]*tmp2-dz[i]*tmp3-0.5*dx[i]*dx[i]*tmp4-0.5*dy[i]*dy[i]*tmp5-0.5*dz[i]*dz[i]*tmp6-dx[i]*dy[i]*tmp7-dx[i]*dz[i]*tmp8-dy[i]*dz[i]*tmp9);
                tmpu[i]=(U_raw[ijk_p]-dx[i]*tmp1_u-dy[i]*tmp2_u-dz[i]*tmp3_u-0.5*dx[i]*dx[i]*tmp4_u
                         -0.5*dy[i]*dy[i]*tmp5_u-0.5*dz[i]*dz[i]*tmp6_u-dx[i]*dy[i]*tmp7_u
                         -dx[i]*dz[i]*tmp8_u-dy[i]*dz[i]*tmp9_u)/dr;
                tmpv[i]=(V_raw[ijk_p]-dx[i]*tmp1_v-dy[i]*tmp2_v-dz[i]*tmp3_v-0.5*dx[i]*dx[i]*tmp4_v
                         -0.5*dy[i]*dy[i]*tmp5_v-0.5*dz[i]*dz[i]*tmp6_v-dx[i]*dy[i]*tmp7_v
                         -dx[i]*dz[i]*tmp8_v-dy[i]*dz[i]*tmp9_v)/dr;
                tmpw[i]=(W_raw[ijk_p]-dx[i]*tmp1_w-dy[i]*tmp2_w-dz[i]*tmp3_w-0.5*dx[i]*dx[i]*tmp4_w
                         -0.5*dy[i]*dy[i]*tmp5_w-0.5*dz[i]*dz[i]*tmp6_w-dx[i]*dy[i]*tmp7_w
                         -dx[i]*dz[i]*tmp8_w-dy[i]*dz[i]*tmp9_w)/dr;
                tmpp[i]=(P_raw[ijk_p]-dx[i]*tmp1_p-dy[i]*tmp2_p-dz[i]*tmp3_p-0.5*dx[i]*dx[i]*tmp4_p
                         -0.5*dy[i]*dy[i]*tmp5_p-0.5*dz[i]*dz[i]*tmp6_p-dx[i]*dy[i]*tmp7_p
                         -dx[i]*dz[i]*tmp8_p-dy[i]*dz[i]*tmp9_p)/dr;
                
            }
            
            U_raw[ijk] = (drp[1]*tmpu[0]+drp[0]*tmpu[1])/(drp[0]+drp[1]);
            V_raw[ijk] = (drp[1]*tmpv[0]+drp[0]*tmpv[1])/(drp[0]+drp[1]);
            W_raw[ijk] = (drp[1]*tmpw[0]+drp[0]*tmpw[1])/(drp[0]+drp[1]);
            P_raw[ijk] = (drp[1]*tmpp[0]+drp[0]*tmpp[1])/(drp[0]+drp[1]);
            U_old_raw[ijk] = U_raw[ijk];
            V_old_raw[ijk] = V_raw[ijk];
            W_old_raw[ijk] = W_raw[ijk];
            P_old_raw[ijk] = P_raw[ijk];
        }
    }
};

void LINKLIST :: Update_point(MESH& mesh, MESH_CARTESIAN& cartesian, FLOW_FIELD& flow_field)
{
    thrust::for_each( LINKLISTPOINT.begin(), LINKLISTPOINT.end(), is_update( mesh, cartesian, flow_field ) );
}