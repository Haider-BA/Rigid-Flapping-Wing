#include "hip/hip_runtime.h"
#include "../include/mesh.h"
#include <thrust/adjacent_difference.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <iostream>
#include <fstream>

struct type_functor : public thrust::unary_function<int,int>
{
    const int IPOINT, JPOINT, KPOINT;
    
    type_functor(int _IPOINT, int _JPOINT, int _KPOINT) : IPOINT(_IPOINT), JPOINT(_JPOINT), KPOINT(_KPOINT) {}
    
    __device__
    int operator() (int INDEX) const
    {
	const int i = INDEX / (JPOINT * KPOINT);
	const int j = (INDEX % (JPOINT * KPOINT)) / KPOINT;
	const int k = (INDEX % (JPOINT * KPOINT)) % KPOINT;
	
	if( (i == 0) || (j == 0) || (k == 0) || (i == (IPOINT-1)) || (j == (JPOINT-1)) || (k == (KPOINT-1)) )
	{
	    if( ((i==0||i==IPOINT-1)&&(j==0)&&(0<=k)&&(k<=KPOINT-1))||
		((i==0||i==IPOINT-1)&&(j==JPOINT-1)&&(0<=k)&&(k<=KPOINT-1))||
		((i==0||i==IPOINT-1)&&(1<=j)&&(j<=JPOINT-2)&&(k==0))||
		((i==0||i==IPOINT-1)&&(1<=j)&&(j<=JPOINT-2)&&(k==KPOINT-1))||
		((1<=i)&&(i<=IPOINT-2)&&(j==0)&&(k==0))||
		((1<=i)&&(i<=IPOINT-2)&&(j==0)&&(k==KPOINT-1))||
		((1<=i)&&(i<=IPOINT-2)&&(j==JPOINT-1)&&(k==0))||
		((1<=i)&&(i<=IPOINT-2)&&(j==JPOINT-1)&&(k==KPOINT-1)) )
	    {
		return 6;
	    }
	    else
	    {
		return 2;
	    }
	}
	else
	{
	    return 1;
	}
    }
};


MESH_CARTESIAN :: MESH_CARTESIAN()
{
    std::cout << "Cartesian mesh is being initialized......";
	
    double Xstart, Xend, Ystart, Yend, Zstart, Zend;
    int Xsegment, Ysegment, Zsegment, XPoint = 1, YPoint = 1, ZPoint = 1;
        
    std::ifstream read;
    read.open("../flapping/cart.dat");
    if(!read.is_open()) std::cout << "Cart.dat is unable to open." << std::endl;
    read >> IPOINT >> JPOINT >> KPOINT;
    POINT_CARTESIAN = IPOINT * JPOINT * KPOINT;
    
    read >> Xstart >> Xsegment;
    double Xsegmentlength[Xsegment], Xratio[Xsegment];
    int Xsegmentpoint[Xsegment];
    Xend = Xstart;
    for(int i = 0; i < Xsegment; i++)
    {
	read >> Xsegmentlength[i] >> Xratio[i] >> Xsegmentpoint[i];
	Xend = Xend + Xsegmentlength[i];
	XPoint = XPoint + Xsegmentpoint[i];
    }
    if(XPoint != IPOINT) std::cout << "Setting of Cartesian mesh is wrong: X_direction." << std::endl;

    read >> Ystart >> Ysegment;
    double Ysegmentlength[Ysegment], Yratio[Ysegment];
    int Ysegmentpoint[Ysegment];
    Yend = Ystart;
    for(int i = 0; i < Ysegment; i++)
    {
	read >> Ysegmentlength[i] >> Yratio[i] >> Ysegmentpoint[i];
	Yend = Yend + Ysegmentlength[i];
	YPoint = YPoint + Ysegmentpoint[i];
    }
    if(YPoint != JPOINT) std::cout << "Setting of Cartesian mesh is wrong: Y_direction" << std::endl;
	
    read >> Zstart >> Zsegment;
    double Zsegmentlength[Zsegment], Zratio[Zsegment];
    int Zsegmentpoint[Zsegment];
    Zend = Zstart;
    for(int i = 0; i < Zsegment; i++)
    {
	read >> Zsegmentlength[i] >> Zratio[i] >> Zsegmentpoint[i];
	Zend = Zend + Zsegmentlength[i];
	ZPoint = ZPoint + Zsegmentpoint[i];
    }
    if(ZPoint != KPOINT) std::cout << "Setting of Cartesian mesh is wrong: Z_direction" << std::endl;
        
    read.close();
	
    XYZ[0].resize(IPOINT);
    XYZ[1].resize(JPOINT);
    XYZ[2].resize(KPOINT);
    
    DELTA[0].resize(IPOINT);
    DELTA[1].resize(JPOINT);
    DELTA[2].resize(KPOINT);
    
    VELOCITY.resize(3);
    ACCELERATION.resize(3);
    
    IINDEX.resize(IPOINT);
    JINDEX.resize(JPOINT);
    
    for(int i = 0; i < 3; i++) POINTTYPE[i].resize(POINT_CARTESIAN);
    
    COUNTTYPE.resize(POINT_CARTESIAN);  /*Need Initialization*/
    
    double tempX[IPOINT], tempY[JPOINT], tempZ[KPOINT];
    Div(Xstart, Xsegment, Xsegmentlength, Xratio, Xsegmentpoint, IPOINT, tempX);
    Div(Ystart, Ysegment, Ysegmentlength, Yratio, Ysegmentpoint, JPOINT, tempY);
    Div(Zstart, Zsegment, Zsegmentlength, Zratio, Zsegmentpoint, KPOINT, tempZ);
    thrust::copy(tempX, tempX+IPOINT, XYZ[0].begin());
    thrust::copy(tempY, tempY+JPOINT, XYZ[1].begin());
    thrust::copy(tempZ, tempZ+KPOINT, XYZ[2].begin());
    thrust::adjacent_difference(XYZ[0].begin(), XYZ[0].end(), DELTA[0].begin()); DELTA[0][0]=0;
    thrust::adjacent_difference(XYZ[1].begin(), XYZ[1].end(), DELTA[1].begin()); DELTA[1][0]=0;
    thrust::adjacent_difference(XYZ[2].begin(), XYZ[2].end(), DELTA[2].begin()); DELTA[2][0]=0;
    
    for(int i = 0; i < 3; i++)
    {
        VELOCITY[i]     = 0;
        ACCELERATION[i] = 0;
    }

    XMIN = XYZ[0][0]; XMAX = XYZ[0][IPOINT-1];
    YMIN = XYZ[1][0]; YMAX = XYZ[1][JPOINT-1];
    ZMIN = XYZ[2][0]; ZMAX = XYZ[2][KPOINT-1];
    MESHSIZE = pow( ( (XMAX-XMIN)/(IPOINT-1) * (YMAX-YMIN)/(JPOINT-1) * (ZMAX-ZMIN)/(KPOINT-1) ), 1.0/3.0 );
    SAFEDISTANCE = 0.1;

    thrust::sequence(IINDEX.begin(), IINDEX.end(), 0, KPOINT*JPOINT);
    thrust::sequence(JINDEX.begin(), JINDEX.end(), 0, KPOINT);
    
    thrust::transform(thrust::make_counting_iterator(0), 
                      thrust::make_counting_iterator(POINT_CARTESIAN),
                      POINTTYPE[0].begin(),
                      type_functor(IPOINT,JPOINT,KPOINT));
    thrust::copy(POINTTYPE[0].begin(), POINTTYPE[0].end(), POINTTYPE[1].begin());
    thrust::copy(POINTTYPE[0].begin(), POINTTYPE[0].end(), POINTTYPE[2].begin());
    
    
    
    
    
    
    std::cout << "Initialization done." << std::endl;
}


MESH_CARTESIAN :: ~MESH_CARTESIAN()
{
    std::cout << "Cartesian mesh is being deleted." << std::endl;
}
    

void MESH_CARTESIAN :: Div(double start, int segment, double *segmentlength, double *ratio, int *segmentpoint, int POINT, double *temp)
{
    double firstsegment;
    int offset = 1; 
    
    temp[0] = start;
    
    for(int i = 0; i < segment; i++)
    {
	if ( fabs(ratio[i] - 1 ) < 1e-4 )
	{
	    firstsegment = segmentlength[i] / segmentpoint[i];
	    
	    for (int j = 0; j < segmentpoint[i]; j++)
	    {
		start = start + firstsegment;
		
		temp[offset + j] = start;
	    }
	    
	    offset = offset + segmentpoint[i];
	}
	else
	{
	    firstsegment = segmentlength[i] * (1 - ratio[i]) / (1 - pow(ratio[i] , segmentpoint[i]));
	    
	    for (int j = 0; j < segmentpoint[i]; j++)
	    {
		start = start + firstsegment * pow(ratio[i], j);
		
		temp[offset + j] = start;
	    }
	    
	    offset = offset + segmentpoint[i];
	}
    }
}

struct is_inner
{
    __device__
    bool operator()(const int x)
    {
	return ( x == 1 );
    }
};

struct is_outer
{
    __device__
    bool operator()(const int x)
    {
	return ( x == 2 );
    }
};

MESH_LESS :: MESH_LESS( REF_FRAME& global )
{
    std::cout << "Meshless is being initialized......";

    POINT_MESSLESS = 0, INNER_POINT = 0, OUTER_POINT = 0;
    OFFSET = new int [global.obj_number];
    INNER_POINT_OFFSET = new int [global.obj_number];
    OUTER_POINT_OFFSET = new int [global.obj_number];
   
    for ( int i = 0; i < global.obj_number; i++ )
    {
	OFFSET[i] = POINT_MESSLESS;
	INNER_POINT_OFFSET[i] = INNER_POINT;
	OUTER_POINT_OFFSET[i] = OUTER_POINT;
	POINT_MESSLESS = POINT_MESSLESS + global.rigid_body[i]->POINT_NUMBER;
	INNER_POINT = INNER_POINT + global.rigid_body[i]->INNER_POINT_NUMBER;
	OUTER_POINT = OUTER_POINT + global.rigid_body[i]->OUTER_POINT_NUMBER;
	
    }
	
    for( int i = 0; i < 3; i++ )
    {
	POSITION[i].resize(POINT_MESSLESS);
	VELOCITY[i].resize(POINT_MESSLESS);
	ACCELERATION[i].resize(POINT_MESSLESS);
	INNER_NODE_INDEX.resize(INNER_POINT);
	OUTER_NODE_INDEX.resize(OUTER_POINT);
	SURFACE_ELE_AREA.resize(INNER_POINT);
    }
	
    std::cout<<"OUTER_POINT is "<<OUTER_POINT<<std::endl;
    for( int i = 0; i < global.obj_number; i++ )
    {
	for( int j = 0; j < 3; j++ )
	{
	    thrust::copy(global.rigid_body[i]->XYZ.row(j).begin(), global.rigid_body[i]->XYZ.row(j).end(), POSITION[j].begin() + OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->UVW.row(j).begin(), global.rigid_body[i]->UVW.row(j).end(), VELOCITY[j].begin() + OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->ACC.row(j).begin(), global.rigid_body[i]->ACC.row(j).end(), ACCELERATION[j].begin() + OFFSET[i]);
	}
	
	thrust::copy_if(thrust::make_counting_iterator(OFFSET[i]), 
			thrust::make_counting_iterator(OFFSET[i] + global.rigid_body[i]->POINT_NUMBER),
			global.rigid_body[i]->INNERMARK.begin(),
			INNER_NODE_INDEX.begin() + INNER_POINT_OFFSET[i],
			is_inner());
	
	thrust::copy_if(thrust::make_counting_iterator(OFFSET[i]),
			thrust::make_counting_iterator(OFFSET[i] + global.rigid_body[i]->POINT_NUMBER),
			global.rigid_body[i]->OUTERMARK.begin(),
			OUTER_NODE_INDEX.begin() + OUTER_POINT_OFFSET[i],
			is_outer());
	
	thrust::copy_if(global.rigid_body[i]->AREA.begin(),
			global.rigid_body[i]->AREA.end(),
			global.rigid_body[i]->INNERMARK.begin(),
			SURFACE_ELE_AREA.begin() + INNER_POINT_OFFSET[i],
			is_inner());
    }

    std::cout << "Initialization done." << std::endl;
}


MESH_LESS :: ~MESH_LESS()
{
    delete[] OFFSET;
    std::cout << "Meshless is being deleted." << std::endl;
}


MESH :: MESH( MESH_CARTESIAN& Cartesian, MESH_LESS& Meshless )
{
    std::cout << "Mesh is being initialized......";
    
    cartesian = &Cartesian;
    meshless  = &Meshless;

    IPOINT = cartesian->IPOINT;
    JPOINT = cartesian->JPOINT;
    KPOINT = cartesian->KPOINT;
    POINT_CARTESIAN = cartesian->POINT_CARTESIAN;
    POINT_MESSLESS = meshless->POINT_MESSLESS;
    POINT_ALL = POINT_CARTESIAN + POINT_MESSLESS;
    
    XYZ.IPOINT          = IPOINT;
    XYZ.JPOINT          = JPOINT;
    XYZ.KPOINT          = KPOINT;
    XYZ.POINT_CARTESIAN = POINT_CARTESIAN;
    XYZ.POINT_MESSLESS  = POINT_MESSLESS;
    XYZ.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
    
    UVW.IPOINT          = IPOINT;
    UVW.JPOINT          = JPOINT;
    UVW.KPOINT          = KPOINT;
    UVW.POINT_CARTESIAN = POINT_CARTESIAN;
    UVW.POINT_MESSLESS  = POINT_MESSLESS;
    UVW.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
    
    ACC.IPOINT          = IPOINT;
    ACC.JPOINT          = JPOINT;
    ACC.KPOINT          = KPOINT;
    ACC.POINT_CARTESIAN = POINT_CARTESIAN;
    ACC.POINT_MESSLESS  = POINT_MESSLESS;
    ACC.POINT_ALL       = POINT_CARTESIAN + POINT_MESSLESS;
        
    for( int s = 0; s < 3; s++ )
    {
        XYZ.XYZ_raw[s]                   = thrust::raw_pointer_cast( cartesian->XYZ[s].data() );
        XYZ.POSITION_MESHLESS_raw[s]     = thrust::raw_pointer_cast( meshless->POSITION[s].data() );
        
        UVW.VELOCITY_MESHLESS_raw[s]     = thrust::raw_pointer_cast( meshless->VELOCITY[s].data() );
        
        ACC.ACCELERATION_MESHLESS_raw[s] = thrust::raw_pointer_cast( meshless->ACCELERATION[s].data() );
    }
    UVW.VELOCITY_CARTESIAN_raw     = thrust::raw_pointer_cast( cartesian->VELOCITY.data() );
    ACC.ACCELERATION_CARTESIAN_raw = thrust::raw_pointer_cast( cartesian->ACCELERATION.data() );
    
    std::cout << "Initialization done." << std::endl;
}

MESH :: ~MESH()
{
    std::cout << "Mesh is being deleted." << std::endl;
}

void MESH :: UPDATE_MESHLESS( REF_FRAME& global )
{
    for( int i = 0; i < global.obj_number; i++ )
    {
	for( int j = 0; j < 3; j++ )
	{
	    thrust::copy(global.rigid_body[i]->XYZ.row(j).begin(), global.rigid_body[i]->XYZ.row(j).end(), meshless->POSITION[j].begin() + meshless->OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->UVW.row(j).begin(), global.rigid_body[i]->UVW.row(j).end(), meshless->VELOCITY[j].begin() + meshless->OFFSET[i]);
	    thrust::copy(global.rigid_body[i]->ACC.row(j).begin(), global.rigid_body[i]->ACC.row(j).end(), meshless->ACCELERATION[j].begin() + meshless->OFFSET[i]);
	}
    }
}

struct key_functor_SEARCH_TYPE0 : public thrust :: unary_function<int, int>
{
    const int ONI_SIZE;
    
    key_functor_SEARCH_TYPE0(int _ONI_SIZE) : ONI_SIZE(_ONI_SIZE) {}
    
    __device__
    int operator()(int x) { return x/ONI_SIZE; }
};

typedef thrust::tuple<int, int, int, int, double> LOCATE_POINT;
/*
typedef struct LOCATE_POINT
{
    int    I;
    int    J;
    int    K;
    int    IM;
    double DISTANCE;
}LOCATE_POINT;
*/

struct functor_LOCATE_POINT : public thrust::unary_function<int, LOCATE_POINT>
{
    int XLENGTH, YLENGTH, ZLENGTH, ISTART, JSTART, KSTART, MLENGTH;
    MESH_CARTESIAN   *cartesian;
    MESH_LESS        *meshless;
    int              *OUTER_NODE_INDEX_raw;
    double           *cartesian_XYZ_raw[3];
    double           *meshless_XYZ_raw[3];
    
    
    functor_LOCATE_POINT(int _XLENGTH, int _YLENGTH, int _ZLENGTH, 
			 int _ISTART, int _JSTART, int _KSTART, int _MLENGTH, 
			 MESH_CARTESIAN *Cartesian, MESH_LESS *Meshless)
    {
	XLENGTH = _XLENGTH; YLENGTH = _YLENGTH; ZLENGTH = _ZLENGTH;
	ISTART = _ISTART; JSTART = _JSTART; KSTART = _KSTART; MLENGTH = _MLENGTH;
	cartesian = Cartesian; meshless = Meshless;
	
	OUTER_NODE_INDEX_raw = thrust::raw_pointer_cast( meshless->OUTER_NODE_INDEX.data() );
	
	for(int s = 0; s < 3; s++)
	{
	    cartesian_XYZ_raw[s] = thrust::raw_pointer_cast( cartesian->XYZ[s].data() );
	    meshless_XYZ_raw[s]  = thrust::raw_pointer_cast( meshless->POSITION[s].data() );
	}
    }
    
    __device__
    LOCATE_POINT operator() (int x) const
    {
	LOCATE_POINT locate_point;
	
	const int im = x % MLENGTH;
	const int i  = (x / MLENGTH) / (YLENGTH * ZLENGTH) + ISTART;
	const int j  = ( (x / MLENGTH) % (YLENGTH * ZLENGTH) ) / ZLENGTH + JSTART;
	const int k  = ( (x / MLENGTH) % (YLENGTH * ZLENGTH) ) % ZLENGTH + KSTART;
	
	double x1,x2,y1,y2,z1,z2;
//	int ONI = meshless->OUTER_NODE_INDEX[im];
	int ONI = OUTER_NODE_INDEX_raw[im];
	
//	x1 = cartesian->XYZ[0][i]; y1 = cartesian->XYZ[1][j]; z1 = cartesian->XYZ[2][k];
//	x2 = meshless->POSITION[0][ONI]; y2 = meshless->POSITION[1][ONI]; z2 = meshless->POSITION[2][ONI];
	
	x1 = cartesian_XYZ_raw[0][i]; y1 = cartesian_XYZ_raw[1][j]; z1 = cartesian_XYZ_raw[2][k];
	x2 = meshless_XYZ_raw[0][ONI]; y2 = meshless_XYZ_raw[1][ONI]; z2 = meshless_XYZ_raw[2][ONI];
	
	const double d = sqrt( (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2) + (z1-z2)*(z1-z2) );
	
	thrust::get<0>(locate_point) = i;
	thrust::get<1>(locate_point) = j;
	thrust::get<2>(locate_point) = k;
	thrust::get<3>(locate_point) = im;
	thrust::get<4>(locate_point) = d;
	
	return locate_point;
/*
	locate_point.I = i;
	locate_point.J = j;
	locate_point.K = k;
	locate_point.IM = im;
	locate_point.DISTANCE = d;
	return locate_point;
*/
    }
};

struct nearest_point : public thrust::binary_function<LOCATE_POINT, LOCATE_POINT, LOCATE_POINT>
{
    __device__
    LOCATE_POINT operator()(LOCATE_POINT x, LOCATE_POINT y)
    {
	if( thrust::get<4>(x) < thrust::get<4>(y) ) return x;
//	if( x.DISTANCE < y.DISTANCE ) return x;
	else return y;
    }
};


void MESH :: SEARCH_TYPE0()
{
    double Xmax,Ymax,Zmax,Xmin,Ymin,Zmin;
    int Istart, Iend, Jstart,Jend, Kstart, Kend;
    std::cout<<"Reach here 0"<<std::endl;
    thrust::pair<thrust::device_vector<double>::iterator, thrust::device_vector<double>::iterator> minmax;
    
    minmax = thrust::minmax_element( meshless->POSITION[0].begin(), meshless->POSITION[0].end() );
    Xmin = *minmax.first; Xmax = *minmax.second;
    
    minmax = thrust::minmax_element( meshless->POSITION[1].begin(), meshless->POSITION[1].end() );
    Ymin = *minmax.first; Ymax = *minmax.second;
    
    minmax = thrust::minmax_element( meshless->POSITION[2].begin(), meshless->POSITION[2].end() );
    Zmin = *minmax.first; Zmax = *minmax.second;
    std::cout<<"Reach here 0.1"<<std::endl;
    if( Xmin <= ( cartesian->XMIN + cartesian->SAFEDISTANCE ) || Xmax >= ( cartesian->XMAX - cartesian->SAFEDISTANCE ) ||
        Ymin <= ( cartesian->YMIN + cartesian->SAFEDISTANCE ) || Ymax >= ( cartesian->YMAX - cartesian->SAFEDISTANCE ) ||
        Zmin <= ( cartesian->ZMIN + cartesian->SAFEDISTANCE ) || Zmax >= ( cartesian->ZMAX - cartesian->SAFEDISTANCE ) )
    {
	std::cout<<"Meshless points are out of range! Need bigger box! Code terminate!"<<std::endl;
    }
    std::cout<<"Reach here 0.2"<<std::endl;
    Kstart = int( (Zmin - cartesian->ZMIN)/cartesian->MESHSIZE ) - 1;
    Kend   = int( (Zmax - cartesian->ZMIN)/cartesian->MESHSIZE ) + 2;
    Jstart = int( (Ymin - cartesian->YMIN)/cartesian->MESHSIZE ) - 1;
    Jend   = int( (Ymax - cartesian->YMIN)/cartesian->MESHSIZE ) + 2;
    Istart = int( (Xmin - cartesian->XMIN)/cartesian->MESHSIZE ) - 1;
    Iend   = int( (Xmax - cartesian->XMIN)/cartesian->MESHSIZE ) + 2;
    std::cout<<"Reach here 0.24"<<std::endl;
    thrust::host_vector<LOCATE_POINT> ijk_im_d_host( (Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1)*(meshless->OUTER_POINT) );
    //(Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1)*meshless->OUTER_POINT
//    cusp::array1d<LOCATE_POINT,cusp::host_memory> ijk_im_d_host;
    std::cout<<"Reach here 0.25"<<std::endl;
//    ijk_im_d_host.resize((Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1)*meshless->OUTER_POINT);
    std::cout<<"Reach here 0.26"<<std::endl;
    cusp::array1d<LOCATE_POINT,cusp::device_memory> ijk_im_d_device = ijk_im_d_host;
    
    std::cout<<"Reach here 0.3"<<std::endl;
    thrust::device_vector<LOCATE_POINT> ijk_im_d_selected((Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1));
    
    std::cout<<"Reach here 1"<<std::endl;
    
    thrust::transform( thrust::make_counting_iterator(0), 
		       thrust::make_counting_iterator( (Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1)*meshless->OUTER_POINT ),
		       ijk_im_d_device.begin(),
		       functor_LOCATE_POINT(Iend-Istart+1, Jend-Jstart+1, Kend-Kstart+1, Istart, Jstart, Kstart, meshless->OUTER_POINT, cartesian, meshless) );
    
    std::cout<<"Reach here 2"<<std::endl;
    
    thrust::equal_to<int> binary_pred;
    thrust::device_vector<int> key_value( (Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1) );
    
    thrust::reduce_by_key( thrust::make_transform_iterator( thrust::make_counting_iterator(0), key_functor_SEARCH_TYPE0( meshless->OUTER_POINT ) ), 
			   thrust::make_transform_iterator( thrust::make_counting_iterator( (Iend-Istart+1)*(Jend-Jstart+1)*(Kend-Kstart+1)*meshless->OUTER_POINT ), key_functor_SEARCH_TYPE0( meshless->OUTER_POINT ) ),
			   ijk_im_d_device.begin(),
//			   thrust::make_discard_iterator(),
			   key_value.begin(),
			   ijk_im_d_selected.begin(),
			   binary_pred,
			   nearest_point() );
    
    
    std::cout<<Istart<<" "<<Iend<<" "<<Jstart<<" "<<Jend<<" "<<Kstart<<" "<<Kend<<std::endl;
    
    
}

